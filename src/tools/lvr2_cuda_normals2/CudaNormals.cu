#include "hip/hip_runtime.h"
// #include <boost/filesystem.hpp>
// #include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

// #include "lvr2/io/ModelFactory.hpp"
// #include "lvr2/util/Timestamp.hpp"
// #include "lvr2/util/IOUtils.hpp"
// #include "Options.hpp"
#include "CudaNormals.cuh"

__global__
void initNormals_kernel(float* normals, size_t num_points)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_points)
    {
        normals[i * 3 + 0] = 1.0;
        normals[i * 3 + 1] = 0.0;
        normals[i * 3 + 2] = 0.0;
    }    
}

void initNormals(float* h_normals, size_t num_points)
{
    int size = num_points * 3;

    float* d_normals;
    hipMalloc(&d_normals, size);

    // Initialize the normals
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock;
    // int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    printf("%d %d \n", threadsPerBlock, blocksPerGrid);

    initNormals_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_normals, num_points);
    // initNormals_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_normals, size);

    hipDeviceSynchronize();

    // Copy the normals back to host
    hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

    hipFree(d_normals);
}


// void setNormals(int argc, char** argv)
// {
//     cuda_normals_2::Options opt(argc, argv);
//     cout << opt << endl;

//     // Get the model
//     ModelPtr model = ModelFactory::readModel(opt.inputFile());

//     // Get the points
//     PointBufferPtr pbuffer = model->m_pointCloud;
//     size_t num_points = model->m_pointCloud->numPoints();

//     floatArr points = pbuffer->getPointArray();
    
//     float* points_raw = &points[0];
    
//     // floatArr normals(new float[num_points * 3]);
//     // Create normals arrays and copy to device
//     int size = num_points * 3;
//     float* h_normals = (float*)malloc(size);

//     float* d_normals;
//     hipMalloc(&d_normals, size);

//     hipMemcpy(d_normals, h_normals, size, hipMemcpyHostToDevice);

//     // Initialize the normals
//     int threadsPerBlock = 256;
//     int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

//     initNormals<<<blocksPerGrid, threadsPerBlock>>>(d_normals, num_points);

//     // Copy the normals back to host
//     hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

//     // Write the normals into the model
//     floatArr normals(new float[size]);

//     for(int i = 0; i < size; i++)
//     {
//         normals[i] = h_normals[i];
//     }

//     pbuffer->setNormalArray(normals, num_points);

//     ModelFactory::saveModel(model, "test.ply");

//     // Free memory
//     hipFree(d_normals);
//     free(h_normals);
// }
