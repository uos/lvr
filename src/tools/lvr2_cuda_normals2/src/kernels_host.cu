#include "kernels_host.h"

#include "LBVHIndex.cuh"

#include <thrust/sort.h>

using namespace lbvh;

// void morton_codes_host(unsigned long long int* h_mortonCodes, float* h_points, int num_points)
// {
//     int size_points = num_points * 3 * sizeof(float);
//     int size_morton = num_points * sizeof(unsigned long long int);

//     int threadsPerBlock = 256;
//     int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock;

//     float* d_points;
//     hipMalloc(&d_points, size_points);
//     hipMemcpy(d_points, h_points, size_points, hipMemcpyHostToDevice);

//     // Get the extent of the point cloud
//     AABB extent = getExtent(h_points, num_points); 

//     unsigned long long int* d_mortonCodes;
//     hipMalloc(&d_mortonCodes, size_morton);
//     hipMemcpy(d_mortonCodes, h_mortonCodes, size_morton, hipMemcpyHostToDevice);
//     morton_code_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_mortonCodes, d_points, num_points, extent);

//     hipDeviceSynchronize();

//     hipMemcpy(h_mortonCodes, d_mortonCodes, size_morton, hipMemcpyDeviceToHost);

//     hipFree(d_points);
//     hipFree(d_mortonCodes);

//     return;

// }

void radix_sort(unsigned long long int* keys, int* values, size_t num_points)
{
    thrust::sort_by_key(keys, keys + num_points, values);

    return;
}

void build_lbvh(float* points, unsigned long num_points)
{
    int leaf_size = 1;
    bool sort_queries = true;
    bool compact = true;
    bool shrinkt_to_fit = true;

    int size_points = num_points * 3 * sizeof(float);
   
    float* d_points;
    hipMalloc(&d_points, size_points);
    hipMemcpy(d_points, points, size_points, hipMemcpyHostToDevice);

    LBVHIndex tree(leaf_size, sort_queries, compact, shrinkt_to_fit);
    
    tree.build(points, num_points);
    
    hipFree(d_points);

    return;
}