#include "kernels_host.h"

#include "LBVHIndex.cuh"

#include <hip/hip_runtime.h>
#include <thrust/sort.h>

using namespace lbvh;

// void morton_codes_host(unsigned long long int* h_mortonCodes, float* h_points, int num_points)
// {
//     int size_points = num_points * 3 * sizeof(float);
//     int size_morton = num_points * sizeof(unsigned long long int);

//     int threadsPerBlock = 256;
//     int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock;

//     float* d_points;
//     hipMalloc(&d_points, size_points);
//     hipMemcpy(d_points, h_points, size_points, hipMemcpyHostToDevice);

//     // Get the extent of the point cloud
//     AABB extent = getExtent(h_points, num_points); 

//     unsigned long long int* d_mortonCodes;
//     hipMalloc(&d_mortonCodes, size_morton);
//     hipMemcpy(d_mortonCodes, h_mortonCodes, size_morton, hipMemcpyHostToDevice);
//     morton_code_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_mortonCodes, d_points, num_points, extent);

//     hipDeviceSynchronize();

//     hipMemcpy(h_mortonCodes, d_mortonCodes, size_morton, hipMemcpyDeviceToHost);

//     hipFree(d_points);
//     hipFree(d_mortonCodes);

//     return;

// }

void radix_sort(unsigned long long int* keys, int* values, size_t num_points)
{
    thrust::sort_by_key(keys, keys + num_points, values);

    return;
}

void build_lbvh(float* points, size_t num_points,
                float* queries, size_t num_queries,
                float* args,
                const char* kernel, const char* kernel_name)
{
    int size_points = num_points * 3 * sizeof(float);

    int leaf_size = 1;
    bool sort_queries = true;
    bool compact = true;
    bool shrink_to_fit = true;

    int K = 1;

    lbvh::LBVHIndex tree(leaf_size, sort_queries, compact, shrink_to_fit);

    std::cout << "Building tree" << std::endl;
    tree.build(points, num_points);
    std::cout << "Done building tree." << std::endl;
    

    // TODO: Don't process the queries here
    tree.process_queries(queries, num_queries, args, points, num_points, 
                        kernel, kernel_name, K);
    

    return;
}