#include "hip/hip_runtime.h"
#include "LBVHIndex.cuh"
#include "lbvh_kernels.cuh"
#include "lbvh.cuh"

#include <stdio.h>
#include <thrust/sort.h>

#include "GPUErrorCheck.h"

using namespace lbvh;

__host__
LBVHIndex::LBVHIndex(int leaf_size, bool sort_queries, bool compact, bool shrink_to_fit)
{
    m_num_objects = 0;
    m_num_nodes = 0;
    m_leaf_size = leaf_size;
    m_sort_queries = sort_queries;
    m_compact = compact;
    m_shrink_to_fit = shrink_to_fit;

}

__host__
void LBVHIndex::build(float* points, size_t num_points)
{
    m_points = points;

    m_num_objects = num_points;
    m_num_nodes = 2 * m_num_objects - 1;

    // initialize AABBs
    AABB* aabbs = (struct AABB*) malloc(sizeof(struct AABB) * num_points);

    // Initial bounding boxes are the points
    for(int i = 0; i < m_num_objects; i ++)
    {
        aabbs[i].min.x = points[3 * i + 0];
        aabbs[i].max.x = points[3 * i + 0];
        aabbs[i].min.y = points[3 * i + 1];
        aabbs[i].max.y = points[3 * i + 1];
        aabbs[i].min.z = points[3 * i + 2];
        aabbs[i].max.z = points[3 * i + 2];
    }
    // Get the extent
    AABB* extent = (struct AABB*) malloc(sizeof(struct AABB)); 
    getExtent(extent, points, m_num_objects);

    AABB* d_extent;
    gpuErrchk(hipMalloc(&d_extent, sizeof(struct AABB)));
    //hipMalloc(&d_extent, sizeof(struct AABB));

    gpuErrchk(hipMemcpy(d_extent, extent, sizeof(struct AABB), hipMemcpyHostToDevice));
    //hipMemcpy(d_extent, extent, sizeof(struct AABB), hipMemcpyHostToDevice);
    
    AABB* d_aabbs;
    gpuErrchk(hipMalloc(&d_aabbs, sizeof(struct AABB) * num_points));
    //hipMalloc(&d_aabbs, sizeof(struct AABB) * num_points);
    
    gpuErrchk(hipMemcpy(d_aabbs, aabbs, sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));
    //hipMemcpy(d_aabbs, aabbs, sizeof(struct AABB) * num_points, hipMemcpyHostToDevice);

    int size_morton = num_points * sizeof(unsigned long long int);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Get the morton codes of the points
    unsigned long long int* d_morton_codes;
    gpuErrchk(hipMalloc(&d_morton_codes, size_morton));
    // hipMalloc(&d_morton_codes, size_morton);

    compute_morton_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_aabbs, d_extent, d_morton_codes, num_points);
    
    gpuErrchk(hipPeekAtLastError());
    
    hipFree(d_aabbs);
    hipFree(d_extent);

    gpuErrchk(hipDeviceSynchronize());
    // hipDeviceSynchronize();

    unsigned long long int* h_morton_codes = (unsigned long long int*)
                    malloc(sizeof(unsigned long long int) * num_points);

    
    hipMemcpy(h_morton_codes, d_morton_codes, size_morton, hipMemcpyDeviceToHost);
    
    hipFree(d_morton_codes);

    // thrust::sort_by_key(keys, keys + num_points, values);
    thrust::sort_by_key(h_morton_codes, h_morton_codes + num_points, 
                        aabbs);
     gpuErrchk(hipPeekAtLastError());

    // for(int i = 0; i < num_points - 1; i++)
    // {
    //     if(h_morton_codes[i] > h_morton_codes[i + 1])
    //     {
    //         printf("Error in sorting \n");
    //         break;
    //     }
    // }
    
    // Create the nodes
    BVHNode* nodes =  (struct BVHNode*) 
                    malloc(sizeof(struct BVHNode) * m_num_nodes); 

    BVHNode* d_nodes;
    gpuErrchk(hipMalloc(&d_nodes, sizeof(struct BVHNode) * m_num_nodes));

    AABB* d_sorted_aabbs;
    gpuErrchk(hipMalloc(&d_sorted_aabbs, 
            sizeof(struct AABB) * num_points));

    gpuErrchk(hipMemcpy(d_sorted_aabbs, aabbs, 
            sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    // Initialize the tree
    initialize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (d_nodes, d_sorted_aabbs, num_points);

    gpuErrchk(hipPeekAtLastError());
    // gpuErrchk(hipFree(0));

    // Construct the tree
    unsigned int* root_node = (unsigned int*)
        malloc(sizeof(unsigned int));
    *root_node = UINT_MAX;

    unsigned int* d_root_node;
    gpuErrchk(hipMalloc(&d_root_node, sizeof(unsigned int)));

    gpuErrchk(hipMemcpy(d_root_node, root_node, sizeof(unsigned int), 
                hipMemcpyHostToDevice));

    unsigned long long int* d_sorted_morton_codes;
    gpuErrchk(hipMalloc(&d_sorted_morton_codes, size_morton));

    gpuErrchk(hipMemcpy(d_sorted_morton_codes, h_morton_codes, 
            size_morton, hipMemcpyHostToDevice));

    construct_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (d_nodes, d_root_node, d_sorted_morton_codes, num_points);

    hipMemcpy(nodes, d_nodes, m_num_nodes * sizeof(BVHNode), 
                hipMemcpyDeviceToHost);

    for(int i = 0; i < m_num_nodes; i++)
    {
        printf("%u\n", nodes[i].range_left);
    }

    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(root_node, d_root_node, 
            sizeof(unsigned int), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_root_node));
    gpuErrchk(hipFree(d_sorted_aabbs));
    gpuErrchk(hipFree(d_sorted_morton_codes));
    gpuErrchk(hipFree(d_nodes));
    
    free(root_node);
    free(nodes);
    free(aabbs);
    free(extent);
    free(h_morton_codes);

    return;
}

// Get the extent of the points 
// (minimum and maximum values in each dimension)
__host__ 
AABB* LBVHIndex::getExtent(AABB* extent, float* points, size_t num_points)
{
    float min_x = INT_MAX;
    float min_y = INT_MAX;
    float min_z = INT_MAX;

    float max_x = INT_MIN;
    float max_y = INT_MIN;
    float max_z = INT_MIN;

    for(int i = 0; i < num_points; i++)
    {
        if(points[3 * i + 0] < min_x)
        {
            min_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] < min_y)
        {
            min_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] < min_z)
        {
            min_z = points[3 * i + 2];
        }

        if(points[3 * i + 0] > max_x)
        {
            max_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] > max_y)
        {
            max_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] > max_z)
        {
            max_z = points[3 * i + 2];
        }
    }
    
    extent->min.x = min_x;
    extent->min.y = min_y;
    extent->min.z = min_z;
    
    extent->max.x = max_x;
    extent->max.y = max_y;
    extent->max.z = max_z;
    
    return extent;
}