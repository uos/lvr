#include "hip/hip_runtime.h"
#include "LBVHIndex.cuh"
#include "lbvh_kernels.cuh"
#include "lbvh.cuh"
#include "normals_kernel.cuh"
#include "aabb.cuh"

#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <exception>
#include <string>
#include <thrust/sort.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "GPUErrorCheck.h"

using namespace lbvh;

// Only for testing
float quadratic_distance(float p1, float p2, float p3, float q1, float q2, float q3)
{
    return (p1 - q1) * (p1 - q1) + (p2 - q2) * (p2 - q2) + (p3 - q3) * (p3 - q3);
}

// Only for testing
void findKNN(int k, float* points, size_t num_points, float* queries, size_t num_queries)
{
    std::cout << "Brute forcing KNN..." << std::endl;
    float neighs[num_queries][k];

    float distances[num_queries][num_points];

    unsigned int indices[num_queries][num_points];

    for(int j = 0; j < num_queries; j++)
    {
        for(int i = 0; i < num_points; i++)
        {
            indices[j][i] = i;
        }

    }

    for(int i = 0; i < num_queries; i++)
    {
        for(int j = 0; j < num_points; j++)
        {
            distances[i][j] = quadratic_distance(
                                    points[3 * j + 0],
                                    points[3 * j + 1],
                                    points[3 * j + 2],
                                    queries[3 * i + 0],
                                    queries[3 * i + 1],
                                    queries[3 * i + 2]);
        }
    }
    for(int i = 0; i < num_queries; i++)
    {
        thrust::sort_by_key(distances[i], distances[i] + num_points, indices[i]);

    }

    for(int i = 0; i < num_queries; i++)
    {
        std::cout << "Query " << i << ": " << std::endl;
        std::cout << "Neighbors: " << std::endl;
        for(int j = 0; j < k; j++){
            std::cout << indices[i][j] << std::endl;
        }
        std::cout << "Distances: " << std::endl;
        for(int j = 0; j < k; j++)
        {
            std::cout << distances[i][j] << std::endl;
        }
    }
}


LBVHIndex::LBVHIndex()
{
    this->m_num_objects = 0;
    this->m_num_nodes = 0;
    this->m_leaf_size = 1;
    this->m_sort_queries = false;
    this->m_compact = false;

    this->m_flip_x = 1000000.0;
    this->m_flip_y = 1000000.0;
    this->m_flip_z = 1000000.0;
    
}

LBVHIndex::LBVHIndex(int leaf_size, bool sort_queries, 
                    bool compact,
                    float flip_x, float flip_y, float flip_z)
{
    this->m_num_objects = 0;
    this->m_num_nodes = 0;
    this->m_leaf_size = leaf_size;
    this->m_sort_queries = sort_queries;
    this->m_compact = compact;

    this->m_flip_x = flip_x;
    this->m_flip_y = flip_y;
    this->m_flip_z = flip_z;
    
}

void LBVHIndex::build(float* points, size_t num_points)
{
    this->m_points = points;

    this->m_num_objects = num_points;
    this->m_num_nodes = 2 * m_num_objects - 1;

    // initialize AABBs
    AABB* aabbs = (struct AABB*) malloc(sizeof(struct AABB) * num_points);

    // Initial bounding boxes are the points
    for(int i = 0; i < m_num_objects; i ++)
    {
        aabbs[i].min.x = points[3 * i + 0];
        aabbs[i].max.x = points[3 * i + 0];
        aabbs[i].min.y = points[3 * i + 1];
        aabbs[i].max.y = points[3 * i + 1];
        aabbs[i].min.z = points[3 * i + 2];
        aabbs[i].max.z = points[3 * i + 2];
    }
    // Get the extent
    AABB* extent = (struct AABB*) malloc(sizeof(struct AABB)); 
    getExtent(extent, points, m_num_objects);

    this->m_extent = extent;

    AABB* d_extent;
    gpuErrchk(hipMalloc(&d_extent, sizeof(struct AABB)));
    gpuErrchk(hipMemcpy(d_extent, extent, sizeof(struct AABB), hipMemcpyHostToDevice));
    
    AABB* d_aabbs;
    gpuErrchk(hipMalloc(&d_aabbs, sizeof(struct AABB) * num_points));
    gpuErrchk(hipMemcpy(d_aabbs, aabbs, sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    int size_morton = num_points * sizeof(unsigned long long int);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Get the morton codes of the points
    unsigned long long int* d_morton_codes;
    gpuErrchk(hipMalloc(&d_morton_codes, size_morton));

    compute_morton_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_aabbs, d_extent, d_morton_codes, num_points);
    
    gpuErrchk(hipPeekAtLastError());
    
    hipFree(d_aabbs);
    hipFree(d_extent);

    gpuErrchk(hipDeviceSynchronize());
    
    unsigned long long int* h_morton_codes = (unsigned long long int*)
                    malloc(sizeof(unsigned long long int) * num_points);

    hipMemcpy(h_morton_codes, d_morton_codes, size_morton, hipMemcpyDeviceToHost);
    
    hipFree(d_morton_codes);


    // Create array of indices with an index for each point
    unsigned int* indices = (unsigned int*)
        malloc(sizeof(unsigned int) * num_points);

    for(int i = 0; i < num_points; i++)
    {
        indices[i] = i;
    }

    // Sort the indices according to the corresponding morton codes
    thrust::sort_by_key(h_morton_codes, h_morton_codes + num_points, 
                        indices);
    
    // Sort the AABBs by the indices
    AABB* sorted_aabbs = (AABB*) malloc(sizeof(AABB) * num_points);
   
    for(int i = 0; i < num_points; i++)
    {
        sorted_aabbs[i] = aabbs[ indices[i] ];
    }

    gpuErrchk(hipPeekAtLastError());

    this->m_sorted_indices = indices;
    
    // Create the nodes
    BVHNode* nodes =  (struct BVHNode*) 
                    malloc(sizeof(struct BVHNode) * m_num_nodes); 

    BVHNode* d_nodes;
    gpuErrchk(hipMalloc(&d_nodes, sizeof(struct BVHNode) * m_num_nodes));

    AABB* d_sorted_aabbs;
    gpuErrchk(hipMalloc(&d_sorted_aabbs, 
            sizeof(struct AABB) * num_points));

    gpuErrchk(hipMemcpy(d_sorted_aabbs, sorted_aabbs, 
            sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    // Initialize the tree
    initialize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (d_nodes, d_sorted_aabbs, num_points);

    gpuErrchk(hipPeekAtLastError());

    // Construct the tree
    unsigned int* root_node = (unsigned int*)
        malloc(sizeof(unsigned int));
    *root_node = UINT_MAX;

    unsigned int* d_root_node;
    gpuErrchk(hipMalloc(&d_root_node, sizeof(unsigned int)));

    gpuErrchk(hipMemcpy(d_root_node, root_node, sizeof(unsigned int), 
                hipMemcpyHostToDevice));

    unsigned long long int* d_sorted_morton_codes;
    gpuErrchk(hipMalloc(&d_sorted_morton_codes, size_morton));

    gpuErrchk(hipMemcpy(d_sorted_morton_codes, h_morton_codes, 
            size_morton, hipMemcpyHostToDevice));

    construct_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (d_nodes, d_root_node, d_sorted_morton_codes, num_points);

    std::cout << "Old nodes number: " << this->m_num_nodes << std::endl;

    // Optimize the tree
    if(this->m_leaf_size > 1)
    {
        std::cout << "Optimizing Tree" << std::endl;
        unsigned int* valid = (unsigned int*)
            malloc(sizeof(unsigned int) * this->m_num_nodes);

        for(int i = 0; i < this->m_num_nodes; i++)
        {
            valid[i] = 1;
        }

        unsigned int* d_valid;
        hipMalloc(&d_valid, sizeof(unsigned int) * this->m_num_nodes);

        hipMemcpy(d_valid, valid, 
            sizeof(unsigned int) * this->m_num_nodes,
            hipMemcpyHostToDevice);

        optimize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_nodes, d_root_node, d_valid, this->m_leaf_size, this->m_num_objects);

        hipMemcpy(valid, d_valid, 
            sizeof(unsigned int) * this->m_num_nodes,
            hipMemcpyDeviceToHost);

        // Compact tree to increase bandwidth
        if(this->m_compact)
        {
            std::cout << "Compacting Tree" << std::endl;
            // Get the cumulative sum of valid, but start with 0
            unsigned int* valid_sums = (unsigned int*)
                malloc(sizeof(unsigned int) * this->m_num_nodes + 1);

            valid_sums[0] = 0;
            for(int i = 1; i < this->m_num_nodes + 1; i++)
            {
                valid_sums[i] = valid_sums[i - 1] + valid[i - 1];
            }

            // Number of the actually used nodes after optimizing
            unsigned int new_node_count = valid_sums[this->m_num_nodes];

            // Calculate the isum parameter
            unsigned int* isum = (unsigned int*)
                malloc(sizeof(unsigned int) * this->m_num_nodes);

            for(int i = 0; i < this->m_num_nodes; i++)
            {
                isum[i] = i - valid_sums[i];
            }
            // Reuse valid space, since it not needed anymore
            unsigned int free_indices_size = isum[new_node_count];

            unsigned int* free = (unsigned int*)
                malloc(sizeof(unsigned int) * free_indices_size);

            free = &valid[0];

            unsigned int* d_valid_sums;
            unsigned int* d_isum;
            unsigned int* d_free;

            hipMalloc(&d_valid_sums, sizeof(unsigned int) * this->m_num_nodes + 1);
            hipMalloc(&d_isum, sizeof(unsigned int) * this->m_num_nodes);
            hipMalloc(&d_free, sizeof(unsigned int) * free_indices_size);

            hipMemcpy(d_valid_sums, valid_sums,
                sizeof(unsigned int) * this->m_num_nodes + 1,
                hipMemcpyHostToDevice);
            hipMemcpy(d_isum, isum,
                sizeof(unsigned int) * this->m_num_nodes,
                hipMemcpyHostToDevice);
            hipMemcpy(d_free, free,
                sizeof(unsigned int) * free_indices_size,
                hipMemcpyHostToDevice);
            
            int threadsPerBlock = 256;
            int blocksPerGrid = (new_node_count + threadsPerBlock - 1) 
                        / threadsPerBlock;

            compute_free_indices_kernel<<<blocksPerGrid, threadsPerBlock>>>
                (d_valid_sums, d_isum, d_free, new_node_count);

            hipMemcpy(valid_sums, d_valid_sums, 
                sizeof(unsigned int) * this->m_num_nodes + 1,
                hipMemcpyDeviceToHost);

            // get the sum of the first object that has to be moved
            unsigned int first_moved = valid_sums[new_node_count];

            threadsPerBlock = 256;
            blocksPerGrid = (this->m_num_nodes + threadsPerBlock - 1) 
                        / threadsPerBlock;

            // self.nodes, root_node, valid_sums_aligned, free, first_moved, new_node_count, self.num_nodes
            compact_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_nodes, d_root_node, d_valid_sums, d_free, first_moved, new_node_count, this->m_num_nodes);

            this->m_num_nodes = new_node_count;
        }

    }
    std::cout << "Done!" << std::endl;
    std::cout << "New nodes number: " << this->m_num_nodes << std::endl;

    hipMemcpy(nodes, d_nodes, m_num_nodes * sizeof(BVHNode), 
                hipMemcpyDeviceToHost);

    this->m_nodes = nodes;

    gpuErrchk(hipMemcpy(root_node, d_root_node, 
            sizeof(unsigned int), hipMemcpyDeviceToHost));
    
    this->m_root_node = root_node[0];
    printf("Root: %u \n", this->m_root_node);

    gpuErrchk(hipFree(d_root_node));
    gpuErrchk(hipFree(d_sorted_aabbs));
    gpuErrchk(hipFree(d_sorted_morton_codes));
    gpuErrchk(hipFree(d_nodes));
    
    // free(root_node);
    // free(nodes);
    // free(aabbs);
    // free(extent);
    // free(h_morton_codes);

    return;
}

#define CUDA_SAFE_CALL(x) \
 do { \
 hipError_t result = x; \
 if (result != hipSuccess) { \
 const char *msg; \
 hipDrvGetErrorName(result, &msg); \
 std::cerr << "\nerror: " #x " failed with error " \
 << msg << '\n'; \
 exit(1); \
 } \
 } while(0)

#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      exit(1);                                                    \
    }                                                             \
  } while(0)


void LBVHIndex::kSearch(
    float* query_points, size_t num_queries,
    int K, 
    unsigned int* n_neighbors_out, unsigned int* indices_out, float* distances_out
)
{
    this->m_radius = FLT_MAX;
    this->process_queries(query_points, num_queries, K, 
        n_neighbors_out, indices_out, distances_out);
}

void LBVHIndex::radiusSearch(
    float* query_points, size_t num_queries,
    int K, int r,
    unsigned int* n_neighbors_out, unsigned int* indices_out, float* distances_out
)
{
    this->m_radius = r;
    this->process_queries(query_points, num_queries, K, 
        n_neighbors_out, indices_out, distances_out);
}

void LBVHIndex::process_queries(
    float* queries_raw, size_t num_queries,
    int K,
    unsigned int* n_neighbors_out, unsigned int* indices_out, float* distances_out
)
{
    // Get the Query Kernel
    std::string kernel_file = "query_knn_kernels.cu";
    std::string kernel_name = "query_knn_kernel";
    std::string kernel_path = "../src/tools/lvr2_cuda_normals2/src/query_knn_kernels.cu";

    std::ifstream in(kernel_path);
    std::string cu_src((std::istreambuf_iterator<char>(in)),
        std::istreambuf_iterator<char>());

    // Get the ptx of the kernel
    std::string ptx_src;

    getPtxFromCuString(ptx_src, kernel_name.c_str(), cu_src.c_str(), NULL, NULL);

    // Init cuda
    hipFree(0);
    
    // Get the cuda module and function
    hipModule_t module;
    hipFunction_t kernel;

    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx_src.c_str(), 0, 0, 0));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernel_name.c_str()));

    // Prepare kernel launch
    BVHNode* d_nodes;
    gpuErrchk( hipMalloc(&d_nodes, sizeof(BVHNode) * this->m_num_nodes) );

    gpuErrchk( hipMemcpy(d_nodes, this->m_nodes, 
            sizeof(BVHNode) * this->m_num_nodes,
            hipMemcpyHostToDevice) );

    //TODO Do this in the kernel
    float3* points3 = (float3*) malloc(sizeof(float3) * this->m_num_objects);
    for(int i = 0; i < this->m_num_objects; i++)
    {
        points3[i].x = this->m_points[3 * i + 0];
        points3[i].y = this->m_points[3 * i + 1];
        points3[i].z = this->m_points[3 * i + 2];
    }

    float3* d_points3;
    gpuErrchk( hipMalloc(&d_points3, sizeof(float3) * this->m_num_objects) );
    gpuErrchk( hipMemcpy(d_points3, points3, 
            sizeof(float3) * this->m_num_objects, 
            hipMemcpyHostToDevice) );
    
    unsigned long long int* d_sorted_indices;
    gpuErrchk( hipMalloc(&d_sorted_indices, sizeof(unsigned int) * this->m_num_objects) );
    gpuErrchk( hipMemcpy(d_sorted_indices, this->m_sorted_indices,
            sizeof(unsigned int) * this->m_num_objects,
            hipMemcpyHostToDevice) );

    //TODO Do this in the kernel
    float3* query_points = (float3*) malloc(sizeof(float3) * num_queries);
    for(int i = 0; i < num_queries; i++)
    {
        query_points[i].x = queries_raw[3 * i + 0];
        query_points[i].y = queries_raw[3 * i + 1];
        query_points[i].z = queries_raw[3 * i + 2];
    }

    float3* d_query_points;
    gpuErrchk( hipMalloc(&d_query_points, sizeof(float3) * num_queries) );
    gpuErrchk( hipMemcpy(d_query_points, query_points,
            sizeof(float3) * num_queries,
            hipMemcpyHostToDevice) );

    unsigned int* sorted_queries = (unsigned int*) 
                malloc(sizeof(unsigned int) * num_queries);

    for(int i = 0; i < num_queries; i++)
    {
        sorted_queries[i] = i;
    }

    // Only for large queries: Sort them in morton order to prevent too much warp divergence on tree traversal
    if(this->m_sort_queries)
    {
        AABB* d_extent;
        gpuErrchk(hipMalloc(&d_extent, sizeof(struct AABB)));
        gpuErrchk(hipMemcpy(d_extent, this->m_extent, sizeof(struct AABB), hipMemcpyHostToDevice));

        unsigned long long int* morton_codes_query =
            (unsigned long long int*)
            malloc(sizeof(unsigned long long int) * num_queries);

        unsigned long long int* d_morton_codes_query;
        hipMalloc(&d_morton_codes_query, 
            sizeof(unsigned long long int) * num_queries);

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;

        compute_morton_points_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_query_points, d_extent, d_morton_codes_query, num_queries);

        hipMemcpy(morton_codes_query, d_morton_codes_query,
            sizeof(unsigned long long int) * num_queries,
            hipMemcpyDeviceToHost);
        
        thrust::sort_by_key(morton_codes_query, morton_codes_query + num_queries, 
                        sorted_queries);
    }

    unsigned int* d_sorted_queries;
    gpuErrchk( hipMalloc(&d_sorted_queries, sizeof(unsigned int) * num_queries) );

    gpuErrchk( hipMemcpy(d_sorted_queries, sorted_queries,
            sizeof(unsigned int) * num_queries,
            hipMemcpyHostToDevice) );

    // Initialise the output arrays
    //TODO Do this in the kernel
    for(int i = 0; i < num_queries * K; i++)
    {
        indices_out[i] = UINT32_MAX;
    }

    unsigned int* d_indices_out;
    gpuErrchk( hipMalloc(&d_indices_out, sizeof(unsigned int) * num_queries * K) );
    gpuErrchk( hipMemcpy(d_indices_out, indices_out,
            sizeof(unsigned int) * num_queries * K,
            hipMemcpyHostToDevice) );

    //TODO Do this in the kernel
    for(int i = 0; i < num_queries * K; i++)
    {
        distances_out[i] = FLT_MAX;
    }

    float* d_distances_out;
    gpuErrchk( hipMalloc(&d_distances_out, sizeof(float) * num_queries * K) );
    gpuErrchk( hipMemcpy(d_distances_out, distances_out,
            sizeof(float) * num_queries * K,
            hipMemcpyHostToDevice) );

    unsigned int* d_n_neighbors_out;
    gpuErrchk( hipMalloc(&d_n_neighbors_out, sizeof(unsigned int) * num_queries) );

    // Gather the arguments
    void *params[] = 
    {
        &d_nodes,
        &d_points3,
        &d_sorted_indices,
        &this->m_root_node,
        &this->m_radius,
        &d_query_points,
        &d_sorted_queries,
        &num_queries,
        &d_indices_out,
        &d_distances_out,
        &d_n_neighbors_out
    };

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Launch the kernel
    CUDA_SAFE_CALL( hipModuleLaunchKernel(kernel, 
        blocksPerGrid, 1, 1,  // grid dim
        threadsPerBlock, 1, 1,    // block dim
        0, NULL,    // shared mem and stream
        params,       // arguments
        0
    ) );      
    
    gpuErrchk( hipMemcpy(indices_out, d_indices_out,
            sizeof(unsigned int) * num_queries * K,
            hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(distances_out, d_distances_out,
            sizeof(float) * num_queries * K,
            hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(n_neighbors_out, d_n_neighbors_out,
            sizeof(unsigned int) * num_queries,
            hipMemcpyDeviceToHost) );

    // findKNN(K, points_raw, num_points, queries_raw, num_queries);

}

 void LBVHIndex::calculate_normals(
    float* normals, size_t num_normals,
    float* queries, size_t num_queries,
    int K,
    unsigned int* n_neighbors_out, unsigned int* indices_out
)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_normals + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Create device memory
    float* d_points;
    gpuErrchk( hipMalloc(&d_points,
        sizeof(float) * 3 * this->m_num_objects));

    float* d_queries;
    gpuErrchk( hipMalloc(&d_queries, 
        sizeof(float) * 3 * num_queries) );

    unsigned int* d_n_neighbors_out;
    gpuErrchk( hipMalloc(&d_n_neighbors_out, 
        sizeof(unsigned int) * num_queries) );

    unsigned int* d_indices_out;
    gpuErrchk( hipMalloc(&d_indices_out, 
        sizeof(unsigned int) * K * num_queries) );

    unsigned int* d_neigh_sum;
    gpuErrchk( hipMalloc(&d_neigh_sum, 
        sizeof(unsigned int) * num_queries) );

    float* d_normals;
    gpuErrchk( hipMalloc(&d_normals, 
        sizeof(float) * 3 * num_normals) );

    // Copy to device
    gpuErrchk( hipMemcpy(d_points, this->m_points,
        sizeof(float) * 3 * this->m_num_objects,
        hipMemcpyHostToDevice));

    gpuErrchk( hipMemcpy(d_queries, queries,
        sizeof(float) * 3 * num_queries,
        hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(d_n_neighbors_out, n_neighbors_out,
        sizeof(unsigned int) * num_queries, 
        hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(d_indices_out, indices_out,
        sizeof(unsigned int) * K * num_queries, 
        hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(d_normals, normals,
        sizeof(float) * 3 * num_normals,
        hipMemcpyHostToDevice) );
    
    // Call the normals kernel
    calculate_normals_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (d_points, d_queries, num_queries, K, d_n_neighbors_out, d_indices_out,
        d_normals, this->m_flip_x, this->m_flip_y, this->m_flip_z);

    hipDeviceSynchronize();

    // Download the normals
    gpuErrchk( hipMemcpy(normals, d_normals,
        sizeof(float) * 3 * num_normals,
        hipMemcpyDeviceToHost) );

}

// Get the extent of the points 
// (minimum and maximum values in each dimension)
AABB* LBVHIndex::getExtent(AABB* extent, float* points, size_t num_points)
{
    float min_x = INT_MAX;
    float min_y = INT_MAX;
    float min_z = INT_MAX;

    float max_x = INT_MIN;
    float max_y = INT_MIN;
    float max_z = INT_MIN;

    for(int i = 0; i < num_points; i++)
    {
        if(points[3 * i + 0] < min_x)
        {
            min_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] < min_y)
        {
            min_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] < min_z)
        {
            min_z = points[3 * i + 2];
        }

        if(points[3 * i + 0] > max_x)
        {
            max_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] > max_y)
        {
            max_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] > max_z)
        {
            max_z = points[3 * i + 2];
        }
    }
    
    extent->min.x = min_x;
    extent->min.y = min_y;
    extent->min.z = min_z;
    
    extent->max.x = max_x;
    extent->max.y = max_y;
    extent->max.z = max_z;
    
    return extent;
}

std::string LBVHIndex::getSampleDir()
{
    // TODO: Don't use hard coded path
    return std::string("/home/till/Develop/src/tools/lvr2_cuda_normals2/src");
}
                         // Rückgabe String // Bsp: square_kernel.cu  // Inhalt d Datei     //Name Programm = NULL
void LBVHIndex::getPtxFromCuString( std::string& ptx, const char* sample_name, const char* cu_source, const char* name, const char** log_string )
{
    // Create program
    hiprtcProgram prog;
    NVRTC_SAFE_CALL( hiprtcCreateProgram( &prog, cu_source, sample_name, 0, NULL, NULL ) );

    // Gather NVRTC options
    std::string cuda_include = std::string("-I") + std::string(CUDA_INCLUDE_DIRS);
    std::vector<const char*> options = {
        "-I/home/till/Develop/src/tools/lvr2_cuda_normals2/include",
        cuda_include.c_str(),
        "-std=c++17",
        "-DK=50"
    };

    const std::string base_dir = getSampleDir();

    // JIT compile CU to PTX
    const hiprtcResult compileRes = hiprtcCompileProgram( prog, (int)options.size(), options.data() );
    
    // Retrieve log output
    size_t log_size = 0;
    NVRTC_SAFE_CALL( hiprtcGetProgramLogSize( prog, &log_size ) );

    char* log = new char[log_size];
    if( log_size > 1 )
    {
        NVRTC_SAFE_CALL( hiprtcGetProgramLog( prog, log ) );
        std::cout << log << std::endl;
    }
    
    if( compileRes != HIPRTC_SUCCESS )
        throw std::runtime_error( "NVRTC Compilation failed.\n");

    // Retrieve PTX code
    size_t ptx_size = 0;
    NVRTC_SAFE_CALL( hiprtcGetCodeSize( prog, &ptx_size ) );
    ptx.resize( ptx_size );
    NVRTC_SAFE_CALL( hiprtcGetCode( prog, &ptx[0] ) );

    // Cleanup
    NVRTC_SAFE_CALL( hiprtcDestroyProgram( &prog ) );
}