
#include <hip/hip_runtime.h>
// Type your code here, or load an example.
extern "C" __global__ void square(int* array, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n)
        array[tid] = array[tid] * array[tid];
}