#include "hip/hip_runtime.h"
#include "query_knn.cuh"

using namespace lvr2;
using namespace lbvh;

namespace lvr2
{
/**
 * @brief   A Cuda kernel that performs a kNN search on the LBVH and
 *          calculates the surface normals by doing an approximated
 *          iterative PCA
 * 
 * @param nodes             Nodes of the LBVH
 * @param points            Points of the dataset
 * @param sorted_indices    Sorted indices of the points
 * @param root_index        Index of the LBVH root node
 * @param max_radius        Maximum radius for radius search
 * @param query_points      Query points for which the normals are calculated
 * @param sorted_queries    Sorted indices of the query points
 * @param num_queries       Number of queries
 * @param normals           Stores the calculated normals
 */
extern "C" __global__ void knn_normals_kernel(
    const BVHNode *nodes,
    const float* __restrict__ points,         
    const unsigned int* __restrict__ sorted_indices,
    const unsigned int root_index,
    const float max_radius,
    const float* __restrict__ query_points,    
    const unsigned int* __restrict__ sorted_queries,
    const unsigned int num_queries, 
    float* normals
)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (tid >= num_queries)
    {
        return;
    }

    double flip_x=1000000.0; 
    double flip_y=1000000.0; 
    double flip_z=1000000.0;
    
    StaticPriorityQueue<float, K> queue(max_radius);

    unsigned int query_idx = sorted_queries[tid];

    float3 query_point =       
    {
        query_points[3 * query_idx + 0],
        query_points[3 * query_idx + 1],
        query_points[3 * query_idx + 2]
    };
    
    query_knn(nodes, points, sorted_indices, root_index, &query_point, queue);
    __syncwarp(); // synchronize the warp before the write operation

    // http://www.ilikebigbits.com/2017_09_25_plane_from_points_2.html

    // We only consider n - 1 neighbors, since the first found neighbor 
    // is the query point itself
    unsigned int n = queue.size();      // only used in for loops
    const double nf = (double) n - 1;   // used for calculation, therefore -1

    if(n < 3)
    {
        // Not enough neighbors
        normals[3 * query_idx + 0] = 0.0f;
        normals[3 * query_idx + 1] = 0.0f;
        normals[3 * query_idx + 2] = 0.0f;
        return; 
    }


    // Get the centroid
    double3 sum = {0.0, 0.0, 0.0};

    for(int i = 1; i < n; i++)
    {
        auto k = queue[i];
        // Coordinates of point which is the i-th neighbor
        sum.x += (double) points[ 3 * k.id + 0] / nf;
        sum.y += (double) points[ 3 * k.id + 1] / nf;
        sum.z += (double) points[ 3 * k.id + 2] / nf;
    }

    // Calculate the covariance matrix
    double xx = 0.0f; double xy = 0.0f; double xz = 0.0f;
    double yy = 0.0f; double yz = 0.0f;
    double zz = 0.0f;

    for(int i = 1; i < n; i++)
    {
        auto k = queue[i];
        double3 r =
        {
            (double) points[ 3 * k.id + 0] - sum.x,     
            (double) points[ 3 * k.id + 1] - sum.y,    
            (double) points[ 3 * k.id + 2] - sum.z      
        };
        xx += r.x * r.x / nf;  
        xy += r.x * r.y / nf;   
        xz += r.x * r.z / nf;   
        yy += r.y * r.y / nf;   
        yz += r.y * r.z / nf;   
        zz += r.z * r.z / nf;   
    }
    
    double3 weighted_dir = {0.0, 0.0, 0.0};
    double3 axis_dir;

    // For x
    double det_x = yy*zz - yz*yz;  

    axis_dir.x = det_x;
    axis_dir.y = xz*yz - xy*zz;
    axis_dir.z = xy*yz - xz*yy;

    double weight = det_x * det_x;

    if(weighted_dir.x * axis_dir.x + weighted_dir.y * axis_dir.y + weighted_dir.z * axis_dir.z < 0.0)
    {
        weight *= -1;
    }
    
    weighted_dir.x += axis_dir.x * weight;
    weighted_dir.y += axis_dir.y * weight;
    weighted_dir.z += axis_dir.z * weight;


    // For y
    double det_y = xx*zz - xz*xz;  

    axis_dir.x = xz*yz - xy*zz;
    axis_dir.y = det_y;
    axis_dir.z = xy*xz - yz*xx;

    weight = det_y * det_y;

    if(weighted_dir.x * axis_dir.x + weighted_dir.y * axis_dir.y + weighted_dir.z * axis_dir.z < 0.0)
    {
        weight *= -1;
    }
    
    weighted_dir.x += axis_dir.x * weight;
    weighted_dir.y += axis_dir.y * weight;
    weighted_dir.z += axis_dir.z * weight;


    // For z
    double det_z = xx*yy - xy*xy; 

    axis_dir.x = xy*yz - xz*yy;
    axis_dir.y = xy*xz - yz*xx;
    axis_dir.z = det_z;

    weight = det_z * det_z;

    if(weighted_dir.x * axis_dir.x + weighted_dir.y * axis_dir.y + weighted_dir.z * axis_dir.z < 0.0)
    {
        weight *= -1;
    }
    
    weighted_dir.x += axis_dir.x * weight;
    weighted_dir.y += axis_dir.y * weight;
    weighted_dir.z += axis_dir.z * weight;

    // Create the normal
    double3 normal = weighted_dir;  

    // Normalize normal
    double mag = sqrt((normal.x * normal.x) + (normal.y * normal.y) + (normal.z * normal.z));  
    normal.x /= mag;
    normal.y /= mag;
    normal.z /= mag;

    // Check if the normals need to be flipped
    double vertex_x = query_points[3 * query_idx + 0];
    double vertex_y = query_points[3 * query_idx + 1];
    double vertex_z = query_points[3 * query_idx + 2];

    double x_dir = flip_x - vertex_x;  
    double y_dir = flip_y - vertex_y;  
    double z_dir = flip_z - vertex_z;   

    double scalar = x_dir * normal.x + y_dir * normal.y + z_dir * normal.z;
    // Set normals to zero if nan or inf values occur
    if(!(scalar <= 0 || scalar >= 0) || isinf(scalar))
    {
        normal.x = 0.0f;
        normal.y = 0.0f;
        normal.z = 0.0f;
    }

    // flip the normals
    if(scalar < 0)
    {
        normal.x = -normal.x;
        normal.y = -normal.y;
        normal.z = -normal.z;
    }

    // Set the normal in the normal array
    normals[3 * query_idx + 0] = (float) normal.x;
    normals[3 * query_idx + 1] = (float) normal.y;
    normals[3 * query_idx + 2] = (float) normal.z;

    return;
}

} // namespace lvr2
