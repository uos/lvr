#include "hip/hip_runtime.h"
#include "lvr2/reconstruction/cuda/LBVHIndex.hpp"
#include "lvr2/reconstruction/cuda/lbvh/lbvh_kernels.cuh"
#include "lvr2/reconstruction/cuda/lbvh/lbvh.cuh"
#include "lvr2/reconstruction/cuda/lbvh/normals_kernel.cuh"
#include "lvr2/reconstruction/cuda/lbvh/aabb.cuh"

#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <exception>
#include <string>
#include <thrust/sort.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "lvr2/reconstruction/cuda/lbvh/GPUErrorCheck.h"

using namespace lvr2;
using namespace lbvh;

/*  TODO
    - Extent device Member?
    - m_d_sorted_indices weg?
    - Delete or change flip_x,... (or add to knn_normals_kernel?)
*/ 


// // Only for testing
// float quadratic_distance(float p1, float p2, float p3, float q1, float q2, float q3)
// {
//     return (p1 - q1) * (p1 - q1) + (p2 - q2) * (p2 - q2) + (p3 - q3) * (p3 - q3);
// }

// // Only for testing
// void findKNN(int k, float* points, size_t num_points, float* queries, size_t num_queries)
// {
//     std::cout << "Brute forcing KNN..." << std::endl;
//     float neighs[num_queries][k];

//     float distances[num_queries][num_points];

//     unsigned int indices[num_queries][num_points];

//     for(int j = 0; j < num_queries; j++)
//     {
//         for(int i = 0; i < num_points; i++)
//         {
//             indices[j][i] = i;
//         }

//     }

//     for(int i = 0; i < num_queries; i++)
//     {
//         for(int j = 0; j < num_points; j++)
//         {
//             distances[i][j] = quadratic_distance(
//                                     points[3 * j + 0],
//                                     points[3 * j + 1],
//                                     points[3 * j + 2],
//                                     queries[3 * i + 0],
//                                     queries[3 * i + 1],
//                                     queries[3 * i + 2]);
//         }
//     }
//     for(int i = 0; i < num_queries; i++)
//     {
//         thrust::sort_by_key(distances[i], distances[i] + num_points, indices[i]);

//     }

//     for(int i = 0; i < num_queries; i++)
//     {
//         std::cout << "Query " << i << ": " << std::endl;
//         std::cout << "Neighbors: " << std::endl;
//         for(int j = 0; j < k; j++){
//             std::cout << indices[i][j] << std::endl;
//         }
//         std::cout << "Distances: " << std::endl;
//         for(int j = 0; j < k; j++)
//         {
//             std::cout << distances[i][j] << std::endl;
//         }
//     }
// }

namespace lvr2
{

LBVHIndex::LBVHIndex()
{
    this->m_num_objects = 0;
    this->m_num_nodes = 0;
    this->m_leaf_size = 1;
    this->m_sort_queries = true;
    this->m_compact = true;

    this->m_flip_x = 1000000.0;
    this->m_flip_y = 1000000.0;
    this->m_flip_z = 1000000.0;
    
}

LBVHIndex::LBVHIndex(
    int leaf_size, 
    bool sort_queries, 
    bool compact,
    float flip_x, 
    float flip_y, 
    float flip_z
)
{
    this->m_num_objects = 0;
    this->m_num_nodes = 0;
    this->m_leaf_size = leaf_size;
    this->m_sort_queries = sort_queries;
    this->m_compact = compact;

    this->m_flip_x = flip_x;
    this->m_flip_y = flip_y;
    this->m_flip_z = flip_z;
    
}

LBVHIndex::~LBVHIndex()
{
    // CPU
    // free(this->m_extent);
    free(this->m_root_node);

    // GPU
    hipFree(this->m_d_points);
    hipFree(this->m_d_sorted_indices);
    hipFree(this->m_d_nodes);
    hipFree(this->m_d_extent);

}

void LBVHIndex::build(float* points, size_t num_points)
{
     // Upload points to GPU
    gpuErrchk( hipMalloc(&this->m_d_points,
        sizeof(float) * 3 * num_points) );
    gpuErrchk( hipMemcpy(this->m_d_points, points,
        sizeof(float) * 3 * num_points,
        hipMemcpyHostToDevice) );

    std::cout << "First Point: " << points[0] << std::endl;
    std::cout << "First Point: " << points[1] << std::endl;
    std::cout << "First Point: " << points[2] << std::endl;
    std::cout << "First Point: " << points[3 * 1000 + 0] << std::endl;
    std::cout << "First Point: " << points[3 * 1000 + 1] << std::endl;
    std::cout << "First Point: " << points[3 * 1000 + 2] << std::endl;

    this->m_num_objects = num_points;
    this->m_num_nodes = 2 * m_num_objects - 1;

    // initialize AABBs
    AABB* aabbs = (struct AABB*) malloc(sizeof(struct AABB) * num_points);

    // Initial bounding boxes are the points
    for(int i = 0; i < m_num_objects; i ++)
    {
        aabbs[i].min.x = points[3 * i + 0];
        aabbs[i].max.x = points[3 * i + 0];
        aabbs[i].min.y = points[3 * i + 1];
        aabbs[i].max.y = points[3 * i + 1];
        aabbs[i].min.z = points[3 * i + 2];
        aabbs[i].max.z = points[3 * i + 2];
    }
    // Get the extent
    AABB* extent = (struct AABB*) malloc(sizeof(struct AABB));
    getExtent(extent, points, m_num_objects);
    
    gpuErrchk(hipMalloc(&this->m_d_extent, sizeof(struct AABB)));
    gpuErrchk(hipMemcpy(this->m_d_extent, extent, sizeof(struct AABB), hipMemcpyHostToDevice));

    AABB* d_aabbs;
    gpuErrchk(hipMalloc(&d_aabbs, sizeof(struct AABB) * num_points));
    gpuErrchk(hipMemcpy(d_aabbs, aabbs, sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    int size_morton = num_points * sizeof(unsigned long long int);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Get the morton codes of the points
    unsigned long long int* d_morton_codes;
    gpuErrchk(hipMalloc(&d_morton_codes, size_morton));

    compute_morton_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_aabbs, this->m_d_extent, d_morton_codes, num_points);
    
    gpuErrchk(hipPeekAtLastError());
    
    // hipFree(d_extent);
    hipFree(d_aabbs);

    // gpuErrchk(hipDeviceSynchronize());
    
    unsigned long long int* morton_codes = (unsigned long long int*)
                    malloc(sizeof(unsigned long long int) * num_points);

    hipMemcpy(morton_codes, d_morton_codes, size_morton, hipMemcpyDeviceToHost);
    
    hipFree(d_morton_codes);

    // Create array of indices with an index for each point
    unsigned int* indices = (unsigned int*)
        malloc(sizeof(unsigned int) * num_points);

    for(int i = 0; i < num_points; i++)
    {
        indices[i] = i;
    }

    // Sort the indices according to the corresponding morton codes
    thrust::sort_by_key(morton_codes, morton_codes + num_points, 
                        indices);
    
    // Sort the AABBs by the indices
    AABB* sorted_aabbs = (AABB*) malloc(sizeof(AABB) * num_points);
   
    for(int i = 0; i < num_points; i++)
    {
        sorted_aabbs[i] = aabbs[indices[i]];
    }

    gpuErrchk(hipPeekAtLastError());

    // TODO nicht 
    hipMalloc(&this->m_d_sorted_indices,
        sizeof(unsigned int) * num_points);
    hipMemcpy(this->m_d_sorted_indices, indices,
        sizeof(unsigned int) * num_points,
        hipMemcpyHostToDevice);
    
    // Create the nodes
    gpuErrchk(hipMalloc(&this->m_d_nodes, sizeof(struct BVHNode) * m_num_nodes));

    AABB* d_sorted_aabbs;
    gpuErrchk(hipMalloc(&d_sorted_aabbs, 
            sizeof(struct AABB) * num_points));

    gpuErrchk(hipMemcpy(d_sorted_aabbs, sorted_aabbs, 
            sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    // Initialize the tree
    initialize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (this->m_d_nodes, d_sorted_aabbs, num_points);

    gpuErrchk(hipPeekAtLastError());

    // Construct the tree
    this->m_root_node = (unsigned int*)
        malloc(sizeof(unsigned int));
    this->m_root_node[0] = UINT_MAX;


    unsigned int* d_root_node;
    gpuErrchk(hipMalloc(&d_root_node, sizeof(unsigned int)));

    gpuErrchk(hipMemcpy(d_root_node, m_root_node, sizeof(unsigned int), 
                hipMemcpyHostToDevice));

    unsigned long long int* d_sorted_morton_codes;
    gpuErrchk(hipMalloc(&d_sorted_morton_codes, size_morton));

    gpuErrchk(hipMemcpy(d_sorted_morton_codes, morton_codes, 
            size_morton, hipMemcpyHostToDevice));

    construct_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (this->m_d_nodes, d_root_node, d_sorted_morton_codes, num_points);

    // Optimize the tree
    if(this->m_leaf_size > 1)
    {
        unsigned int* valid = (unsigned int*)
            malloc(sizeof(unsigned int) * this->m_num_nodes);

        // TODO Initialise this in kernel?
        for(int i = 0; i < this->m_num_nodes; i++)
        {
            valid[i] = 1;
        }

        unsigned int* d_valid;
        hipMalloc(&d_valid, sizeof(unsigned int) * this->m_num_nodes);

        hipMemcpy(d_valid, valid, 
            sizeof(unsigned int) * this->m_num_nodes,
            hipMemcpyHostToDevice);

        optimize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (this->m_d_nodes, d_root_node, d_valid, this->m_leaf_size, this->m_num_objects);

        hipMemcpy(valid, d_valid, 
            sizeof(unsigned int) * this->m_num_nodes,
            hipMemcpyDeviceToHost);

        // Compact tree to increase bandwidth
        if(this->m_compact)
        {
            // Get the cumulative sum of valid, but start with 0
            unsigned int* valid_sums = (unsigned int*)
                malloc(sizeof(unsigned int) * this->m_num_nodes + 1);

            valid_sums[0] = 0;
            for(int i = 1; i < this->m_num_nodes + 1; i++)
            {
                valid_sums[i] = valid_sums[i - 1] + valid[i - 1];
            }

            // Number of the actually used nodes after optimizing
            unsigned int new_node_count = valid_sums[this->m_num_nodes];

            // Calculate the isum parameter
            unsigned int* isum = (unsigned int*)
                malloc(sizeof(unsigned int) * this->m_num_nodes);

            for(int i = 0; i < this->m_num_nodes; i++)
            {
                isum[i] = i - valid_sums[i];
            }
            // Reuse valid space, since its not needed anymore
            unsigned int free_indices_size = isum[new_node_count];

            unsigned int* free_indices = (unsigned int*)
                malloc(sizeof(unsigned int) * free_indices_size);

            free_indices = &valid[0];

            // Upload
            unsigned int* d_valid_sums;
            unsigned int* d_isum;
            unsigned int* d_free_indices;

            hipMalloc(&d_valid_sums, sizeof(unsigned int) * this->m_num_nodes + 1);
            hipMalloc(&d_isum, sizeof(unsigned int) * this->m_num_nodes);
            hipMalloc(&d_free_indices, sizeof(unsigned int) * free_indices_size);

            hipMemcpy(d_valid_sums, valid_sums,
                sizeof(unsigned int) * this->m_num_nodes + 1,
                hipMemcpyHostToDevice);
            hipMemcpy(d_isum, isum,
                sizeof(unsigned int) * this->m_num_nodes,
                hipMemcpyHostToDevice);
            hipMemcpy(d_free_indices, free_indices,
                sizeof(unsigned int) * free_indices_size,
                hipMemcpyHostToDevice);
            
            int threadsPerBlock = 256;
            int blocksPerGrid = (new_node_count + threadsPerBlock - 1) 
                        / threadsPerBlock;

            compute_free_indices_kernel<<<blocksPerGrid, threadsPerBlock>>>
                (d_valid_sums, d_isum, d_free_indices, new_node_count);

            hipMemcpy(valid_sums, d_valid_sums, 
                sizeof(unsigned int) * this->m_num_nodes + 1,
                hipMemcpyDeviceToHost);

            // get the sum of the first object that has to be moved
            unsigned int first_moved = valid_sums[new_node_count];

            threadsPerBlock = 256;
            blocksPerGrid = (this->m_num_nodes + threadsPerBlock - 1) 
                        / threadsPerBlock;

            // self.nodes, root_node, valid_sums_aligned, free, first_moved, new_node_count, self.num_nodes
            compact_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (this->m_d_nodes, d_root_node, d_valid_sums, d_free_indices, first_moved, new_node_count, this->m_num_nodes);

            this->m_num_nodes = new_node_count;
            free(valid_sums);
            free(isum);
            // free(free_indices);

            hipFree(d_valid_sums);
            hipFree(d_isum);
            hipFree(d_free_indices);
        }
        hipFree(d_valid);

        std::cout << "Done with Optimization" << std::endl;
        free(valid);
    }
    
    gpuErrchk(hipMemcpy(m_root_node, d_root_node, 
            sizeof(unsigned int), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_root_node));
    gpuErrchk(hipFree(d_sorted_aabbs));
    gpuErrchk(hipFree(d_sorted_morton_codes));
    

    free(aabbs);
    free(morton_codes);
    free(indices);
    free(sorted_aabbs);

    return;
}

#define CUDA_SAFE_CALL(x) \
 do { \
 hipError_t result = x; \
 if (result != hipSuccess) { \
 const char *msg; \
 hipDrvGetErrorName(result, &msg); \
 std::cerr << "\nerror: " #x " failed with error " \
 << msg << '\n'; \
 exit(1); \
 } \
 } while(0)

#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      exit(1);                                                    \
    }                                                             \
  } while(0)

void LBVHIndex::kSearch(
    float* query_points, 
    size_t num_queries,
    int K, 
    unsigned int* n_neighbors_out, 
    unsigned int* indices_out, 
    float* distances_out
) const
{   
    float radius = FLT_MAX;
    
    this->process_queries(query_points, num_queries, K, radius,
        n_neighbors_out, indices_out, distances_out);
}

void LBVHIndex::kSearch_dev_ptr(
    float* d_query_points, 
    size_t num_queries,
    int K, 
    unsigned int* d_n_neighbors_out, 
    unsigned int* d_indices_out, 
    float* d_distances_out
) const
{
    float radius = FLT_MAX;
    
    this->process_queries_dev_ptr(d_query_points, num_queries, K, radius,
        d_n_neighbors_out, d_indices_out, d_distances_out);
}

void LBVHIndex::radiusSearch(
    float* query_points, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* n_neighbors_out, 
    unsigned int* indices_out, 
    float* distances_out
) const
{   
    this->process_queries(query_points, num_queries, K, r,
        n_neighbors_out, indices_out, distances_out);
}

void LBVHIndex::radiusSearch_dev_ptr(
    float* d_query_points, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* d_n_neighbors_out, 
    unsigned int* d_indices_out, 
    float* d_distances_out
) const
{
    this->process_queries_dev_ptr(d_query_points, num_queries, K, r,
        d_n_neighbors_out, d_indices_out, d_distances_out);
}

void LBVHIndex::process_queries(
    float* queries_raw, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* n_neighbors_out, 
    unsigned int* indices_out, 
    float* distances_out
) const
{
    hipEvent_t start, stop;
    float elapsedTime;

    unsigned int* d_n_neighbors_out; 
    unsigned int* d_indices_out; 
    float* d_distances_out;

    hipEventCreate(&start);
    hipEventRecord(start,0);
    
    // Allocate output buffer
    gpuErrchk( hipMalloc(&d_indices_out, sizeof(unsigned int) * num_queries * K) );
    gpuErrchk( hipMalloc(&d_distances_out, sizeof(float) * num_queries * K) );
    gpuErrchk( hipMalloc(&d_n_neighbors_out, sizeof(unsigned int) * num_queries) );

    // Upload
    float* d_query_points;
    hipMalloc(&d_query_points, sizeof(float) * 3 * num_queries);
    hipMemcpy(d_query_points, queries_raw,
        sizeof(float) * 3 * num_queries,
        hipMemcpyHostToDevice);
    
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Upload Time: " << elapsedTime << std::endl;
    
    // Compute on GPU
    this->process_queries_dev_ptr(
        d_query_points, 
        num_queries,
        K, 
        r,
        d_n_neighbors_out, 
        d_indices_out, 
        d_distances_out
    );

    hipEventCreate(&start);
    hipEventRecord(start,0);
    
    // Download
    gpuErrchk( hipMemcpy(indices_out, d_indices_out,
            sizeof(unsigned int) * num_queries * K,
            hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(distances_out, d_distances_out,
            sizeof(float) * num_queries * K,
            hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(n_neighbors_out, d_n_neighbors_out,
            sizeof(unsigned int) * num_queries,
            hipMemcpyDeviceToHost) );

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Download Time: " << elapsedTime << std::endl;

    hipFree(d_indices_out);
    hipFree(d_distances_out);
    hipFree(d_n_neighbors_out);
    hipFree(d_query_points);
}

void LBVHIndex::process_queries_dev_ptr(
    float* d_query_points, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* d_n_neighbors_out, 
    unsigned int* d_indices_out, 
    float* d_distances_out
) const
{
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);
    
    // Get the Query Kernel
    std::string kernel_file = "query_knn_kernels.cu";
    std::string kernel_name = "query_knn_kernel";
    std::string kernel_dir = std::string(LBVH_KERNEL_DIR);
    std::string kernel_path = kernel_dir + "/" + kernel_file;

    // Read the kernel file
    std::ifstream in(kernel_path);
    std::string cu_src((std::istreambuf_iterator<char>(in)),
        std::istreambuf_iterator<char>());

    // Get the ptx of the kernel
    std::string ptx_src;

    getPtxFromCuString(
        ptx_src, 
        kernel_name.c_str(), 
        cu_src.c_str(), 
        K
    );
    // Init cuda
    hipFree(0);
    
    // Get cuda module and function
    hipModule_t module;
    hipFunction_t kernel;

    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx_src.c_str(), 0, 0, 0));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernel_name.c_str()));

    // Prepare kernel launch
    // TODO Als member? Sonst free()
    unsigned int* sorted_queries = (unsigned int*) 
                malloc(sizeof(unsigned int) * num_queries);

    for(int i = 0; i < num_queries; i++)
    {
        sorted_queries[i] = i;
    }

    // Sort the queries according to their morton codes
    if(this->m_sort_queries)
    {
        // AABB* d_extent;
        // gpuErrchk(hipMalloc(&d_extent, sizeof(struct AABB)));
        // gpuErrchk(hipMemcpy(d_extent, this->m_extent, sizeof(struct AABB), hipMemcpyHostToDevice));

        unsigned long long int* morton_codes_query =
            (unsigned long long int*)
            malloc(sizeof(unsigned long long int) * num_queries);

        unsigned long long int* d_morton_codes_query;
        hipMalloc(&d_morton_codes_query, 
            sizeof(unsigned long long int) * num_queries);

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;

        compute_morton_points_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_query_points, this->m_d_extent, d_morton_codes_query, num_queries);

        hipMemcpy(morton_codes_query, d_morton_codes_query,
            sizeof(unsigned long long int) * num_queries,
            hipMemcpyDeviceToHost);
        
        thrust::sort_by_key(morton_codes_query, morton_codes_query + num_queries, 
                        sorted_queries);
    
        // hipFree(d_extent);
    }

    // Upload
    unsigned int* d_sorted_queries;
    gpuErrchk( hipMalloc(&d_sorted_queries, sizeof(unsigned int) * num_queries) );
    gpuErrchk( hipMemcpy(d_sorted_queries, sorted_queries,
            sizeof(unsigned int) * num_queries,
            hipMemcpyHostToDevice) );

    // TODO Is there a better way to do this?
    BVHNode* d_nodes = const_cast<BVHNode*>(this->m_d_nodes);
    float* d_points = const_cast<float*>(this->m_d_points);
    unsigned int* d_sorted_indices = const_cast<unsigned int*>(this->m_d_sorted_indices);
    unsigned int root_node = this->m_root_node[0];
    float radius = r;

    void *params[] = 
    {
        &d_nodes,
        &d_points,
        &d_sorted_indices,
        &root_node,
        &radius,
        &d_query_points,
        &d_sorted_queries,
        &num_queries,
        &d_indices_out,
        &d_distances_out,
        &d_n_neighbors_out
    };

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;
   
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Upload 2 Time: " << elapsedTime << std::endl;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    // Launch the kernel
    CUDA_SAFE_CALL( hipModuleLaunchKernel(kernel, 
        blocksPerGrid, 1, 1,  // grid dim
        threadsPerBlock, 1, 1,    // block dim
        0, NULL,    // shared mem and stream
        params,       // arguments
        0
    ) );      
    
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "KNN Kernel Time: " << elapsedTime << std::endl;

    hipFree(d_query_points);
    hipFree(d_sorted_queries);

    free(sorted_queries);
    
    return;
}

 void LBVHIndex::calculate_normals(
    float* normals, 
    size_t num_normals,
    float* queries, 
    size_t num_queries,
    int K,
    const unsigned int* n_neighbors_in, 
    const unsigned int* indices_in
)   const
{
    hipEvent_t start, stop;
    float elapsedTime;

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_normals + threadsPerBlock - 1) 
                        / threadsPerBlock;

    hipEventCreate(&start);
    hipEventRecord(start,0);
    

    // Create device memory
    float* d_queries;
    gpuErrchk( hipMalloc(&d_queries, 
        sizeof(float) * 3 * num_queries) );
    
    float* d_normals;
    gpuErrchk( hipMalloc(&d_normals, 
        sizeof(float) * 3 * num_normals) );

    unsigned int* d_n_neighbors_in;
    gpuErrchk( hipMalloc(&d_n_neighbors_in, 
        sizeof(unsigned int) * num_queries) );

    unsigned int* d_indices_in;
    gpuErrchk( hipMalloc(&d_indices_in, 
        sizeof(unsigned int) * K * num_queries) );

    // Upload
    gpuErrchk( hipMemcpy(d_queries, queries,
        sizeof(float) * 3 * num_queries,
        hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_n_neighbors_in, n_neighbors_in,
        sizeof(unsigned int) * num_queries, 
        hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_indices_in, indices_in,
        sizeof(unsigned int) * K * num_queries, 
        hipMemcpyHostToDevice) );
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Upload Time: " << elapsedTime << std::endl;


    hipEventCreate(&start);
    hipEventRecord(start,0);
    
    // Call the normals kernel
    calculate_normals_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (this->m_d_points, d_queries, num_queries, K, d_n_neighbors_in, d_indices_in,
        d_normals, this->m_flip_x, this->m_flip_y, this->m_flip_z);
    
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Normals Kernel Time: " << elapsedTime << std::endl;    
    
    // Download the normals
    hipEventCreate(&start);
    hipEventRecord(start,0);
    
    gpuErrchk( hipMemcpy(normals, d_normals,
        sizeof(float) * 3 * num_normals,
        hipMemcpyDeviceToHost) );

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Download Time: " << elapsedTime << std::endl;
    
    hipFree(d_queries);
    hipFree(d_normals);
    hipFree(d_indices_in);
    hipFree(d_n_neighbors_in);

    bool first = true;
    size_t count = 0;
    for(int i = 0; i < 3 * num_normals; i+=3)
    {
        if( normals[i + 0] == 0.0f && 
            normals[i + 1] == 0.0f && 
            normals[i + 2] == 0.0f)
            {
                if(first)
                {
                    first = false;
                    std::cout << "First uninit normal: " << i << std::endl;
                }
                count++;
            }
    }

    std::cout << "Uninitialised normals: " << count << std::endl;

}
// TODO Make this const?
void LBVHIndex::knn_normals(
    int K,
    float* normals, 
    size_t num_normals
)
{
    float* d_query_points = this->m_d_points;
    size_t num_queries = num_normals;
    
    hipEvent_t start, stop;
    float elapsedTime;

    std::cout << "Loading Kernel..." << std::endl;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    // Get the KNN Normals Kernel
    std::string kernel_file = "knn_normals_kernel.cu";
    std::string kernel_name = "knn_normals_kernel";
    std::string kernel_dir = std::string(LBVH_KERNEL_DIR);
    std::string kernel_path = kernel_dir + "/" + kernel_file;
    
    // Read the kernel file
    std::ifstream in(kernel_path);
    std::string cu_src((std::istreambuf_iterator<char>(in)),
        std::istreambuf_iterator<char>());

    // Get the ptx of the kernel
    std::string ptx_src;

    getPtxFromCuString(
        ptx_src, 
        kernel_name.c_str(), 
        cu_src.c_str(), 
        K
    );

    // Init cuda
    hipFree(0);
    
    // Get the cuda module and function
    hipModule_t module;
    hipFunction_t kernel;

    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx_src.c_str(), 0, 0, 0));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernel_name.c_str()));

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Loading Kernel Time: " << elapsedTime << std::endl;

    std::cout << "Uploading..." << std::endl;
    hipEventCreate(&start);
    hipEventRecord(start,0);

    // Prepare kernel launch
    // TODO Als member
    unsigned int* sorted_queries = (unsigned int*) 
                malloc(sizeof(unsigned int) * num_queries);

    for(int i = 0; i < num_queries; i++)
    {
        sorted_queries[i] = i;
    }

    // TODO Comment in
    // Only for large queries: Sort them in morton order to prevent too much warp divergence on tree traversal
    if(this->m_sort_queries)
    {
        // AABB* d_extent;
        // gpuErrchk(hipMalloc(&d_extent, sizeof(struct AABB)));
        // gpuErrchk(hipMemcpy(d_extent, this->m_extent, 
        //     sizeof(struct AABB), 
        //     hipMemcpyHostToDevice));

        unsigned long long int* morton_codes_query =
            (unsigned long long int*)
            malloc(sizeof(unsigned long long int) * num_queries);

        unsigned long long int* d_morton_codes_query;
        hipMalloc(&d_morton_codes_query, 
            sizeof(unsigned long long int) * num_queries);

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;

        compute_morton_points_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_query_points, this->m_d_extent, d_morton_codes_query, num_queries);

        hipMemcpy(morton_codes_query, d_morton_codes_query,
            sizeof(unsigned long long int) * num_queries,
            hipMemcpyDeviceToHost);

        thrust::sort_by_key(morton_codes_query, morton_codes_query + num_queries, 
                        sorted_queries);

        // hipFree(d_extent);
        hipFree(d_morton_codes_query);
    }

    unsigned int* d_sorted_queries;
    gpuErrchk( hipMalloc(&d_sorted_queries, sizeof(unsigned int) * num_queries) );
    gpuErrchk( hipMemcpy(d_sorted_queries, sorted_queries,
            sizeof(unsigned int) * num_queries,
            hipMemcpyHostToDevice) );

    float* d_normals;
    gpuErrchk( hipMalloc(&d_normals, 
        sizeof(float) * 3 * num_normals) );

    float radius = 30.0f;

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "Upload Time: " << elapsedTime << std::endl;
    
    // Gather the arguments
    void *params[] = 
    {
        &this->m_d_nodes,
        &this->m_d_points,
        &this->m_d_sorted_indices,
        &this->m_root_node[0],
        &radius,
        &d_query_points,
        &d_sorted_queries,
        &num_queries,
        &d_normals
    };

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;
    std::cout << "Launching Kernel..." << std::endl;

    hipEventCreate(&start);
    hipEventRecord(start,0);
    
    // Launch the kernel
    CUDA_SAFE_CALL( hipModuleLaunchKernel(kernel, 
        blocksPerGrid, 1, 1,  // grid dim
        threadsPerBlock, 1, 1,    // block dim
        0, NULL,    // shared mem and stream
        params,       // arguments
        0
    ) );    

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    
    std::cout << "KNN Normals Kernel Time: " << elapsedTime << std::endl;
    
    std::cout << "Downloading..." << std::endl;
    
    hipEventCreate(&start);
    hipEventRecord(start,0);
    

    gpuErrchk( hipMemcpy(normals, d_normals,
        sizeof(float) * 3 * num_normals,
        hipMemcpyDeviceToHost) );  

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    std::cout << "Download Time: " << elapsedTime << std::endl;
    
    hipFree(d_sorted_queries);
    hipFree(d_normals);

    bool first = true;
    size_t count = 0;
    for(int i = 0; i < 3 * num_normals; i+=3)
    {
        if( normals[i + 0] == 0.0f && 
            normals[i + 1] == 0.0f && 
            normals[i + 2] == 0.0f)
            {
                if(first)
                {
                    first = false;
                    std::cout << "First uninit normal: " << i << std::endl;
                }
                count++;
            }
    }

    // int idx = 0;
    // std::cout << "Normal: " << std::endl;
    // for(int i = 1; i < 10; i++)
    // {
    //     std::cout << "x: " << normals[3 * idx + 0 + 3*i] << std::endl;
    //     std::cout << "y: " << normals[3 * idx + 1 + 3*i] << std::endl;
    //     std::cout << "z: " << normals[3 * idx + 2 + 3*i] << std::endl;
    // //     std::cout << "Neigh idx: " << normals[i] << std::endl;

    // }

    // std::cout << "xx: " << normals[3 * idx + 0] << std::endl;
    // std::cout << "xy: " << normals[3 * idx + 1] << std::endl;
    // std::cout << "xz: " << normals[3 * idx + 2] << std::endl;
    // std::cout << "yy: " << normals[3 * idx + 3] << std::endl;
    // std::cout << "yz: " << normals[3 * idx + 4] << std::endl;
    // std::cout << "zz: " << normals[3 * idx + 5] << std::endl;

    std::cout << "Uninitialised normals: " << count << std::endl;
}

// Get the extent of the points 
// (minimum and maximum values in each dimension)
void LBVHIndex::getExtent(
    AABB* extent, 
    float* points, 
    size_t num_points
) const
{
    float min_x = INT_MAX;
    float min_y = INT_MAX;
    float min_z = INT_MAX;

    float max_x = INT_MIN;
    float max_y = INT_MIN;
    float max_z = INT_MIN;

    for(int i = 0; i < num_points; i++)
    {
        if(points[3 * i + 0] < min_x)
        {
            min_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] < min_y)
        {
            min_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] < min_z)
        {
            min_z = points[3 * i + 2];
        }

        if(points[3 * i + 0] > max_x)
        {
            max_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] > max_y)
        {
            max_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] > max_z)
        {
            max_z = points[3 * i + 2];
        }
    }
    
    extent->min.x = min_x;
    extent->min.y = min_y;
    extent->min.z = min_z;
    
    extent->max.x = max_x;
    extent->max.y = max_y;
    extent->max.z = max_z;
    
    return;
}

void LBVHIndex::getPtxFromCuString( 
    std::string& ptx, 
    const char* sample_name, 
    const char* cu_source, 
    int K
) const
{
    // Create program
    hiprtcProgram prog;
    NVRTC_SAFE_CALL( hiprtcCreateProgram( &prog, cu_source, sample_name, 0, NULL, NULL ) );

    std::string K_str = "-DK=" + std::to_string(K); 

    // Gather NVRTC options
    std::string kernel_includes = std::string("-I") + 
        std::string(LBVH_KERNEL_INCLUDES);

    std::string cuda_include = std::string("-I") + 
        std::string(CUDA_INCLUDE_DIRS);
    
    std::vector<const char*> options = {
        kernel_includes.c_str(),
        cuda_include.c_str(),
        "-std=c++17",
        K_str.c_str()
    };

    const std::string base_dir = std::string(LBVH_KERNEL_DIR);

    // JIT compile CU to PTX
    const hiprtcResult compileRes = hiprtcCompileProgram( prog, (int)options.size(), options.data() );
    
    // Retrieve log output
    size_t log_size = 0;
    NVRTC_SAFE_CALL( hiprtcGetProgramLogSize( prog, &log_size ) );

    char* log = new char[log_size];
    if( log_size > 1 )
    {
        NVRTC_SAFE_CALL( hiprtcGetProgramLog( prog, log ) );
        std::cout << log << std::endl;
    }
    
    if( compileRes != HIPRTC_SUCCESS )
        throw std::runtime_error( "NVRTC Compilation failed.\n");

    // Retrieve PTX code
    size_t ptx_size = 0;
    NVRTC_SAFE_CALL( hiprtcGetCodeSize( prog, &ptx_size ) );
    ptx.resize( ptx_size );
    NVRTC_SAFE_CALL( hiprtcGetCode( prog, &ptx[0] ) );

    // Cleanup
    NVRTC_SAFE_CALL( hiprtcDestroyProgram( &prog ) );
}

} // namespace lvr2
