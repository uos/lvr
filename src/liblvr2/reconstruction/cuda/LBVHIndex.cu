#include "hip/hip_runtime.h"
#include "lvr2/reconstruction/cuda/LBVHIndex.hpp"
#include "lvr2/reconstruction/cuda/lbvh/lbvh_kernels.cuh"
#include "lvr2/reconstruction/cuda/lbvh/lbvh.cuh"
#include "lvr2/reconstruction/cuda/lbvh/normals_kernel.cuh"
#include "lvr2/reconstruction/cuda/lbvh/aabb.cuh"

#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <exception>
#include <string>
#include <thrust/sort.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "lvr2/reconstruction/cuda/lbvh/GPUErrorCheck.h"

using namespace lvr2;
using namespace lbvh;

namespace lvr2
{

LBVHIndex::LBVHIndex()
{
    this->m_num_objects = 0;
    this->m_num_nodes = 0;
    this->m_leaf_size = 1;
    this->m_sort_queries = true;
    this->m_compact = true;

    this->m_flip_x = 1000000.0;
    this->m_flip_y = 1000000.0;
    this->m_flip_z = 1000000.0;
    
}

LBVHIndex::LBVHIndex(
    int leaf_size, 
    bool sort_queries, 
    bool compact,
    float flip_x, 
    float flip_y, 
    float flip_z
)
{
    this->m_num_objects = 0;
    this->m_num_nodes = 0;
    this->m_leaf_size = leaf_size;
    this->m_sort_queries = sort_queries;
    this->m_compact = compact;

    this->m_flip_x = flip_x;
    this->m_flip_y = flip_y;
    this->m_flip_z = flip_z;
    
}

LBVHIndex::~LBVHIndex()
{
    // CPU
    free(this->m_root_node);

    // GPU
    hipFree(this->m_d_points);
    hipFree(this->m_d_sorted_indices);
    hipFree(this->m_d_nodes);
    hipFree(this->m_d_extent);

}

void LBVHIndex::build(float* points, size_t num_points)
{
    this->m_num_objects = num_points;
    this->m_num_nodes = 2 * m_num_objects - 1;

    // initialize AABBs
    AABB* aabbs = (struct AABB*) malloc(sizeof(struct AABB) * num_points);

    // Initial bounding boxes are the points
    for(int i = 0; i < m_num_objects; i ++)
    {
        aabbs[i].min.x = points[3 * i + 0];
        aabbs[i].max.x = points[3 * i + 0];
        aabbs[i].min.y = points[3 * i + 1];
        aabbs[i].max.y = points[3 * i + 1];
        aabbs[i].min.z = points[3 * i + 2];
        aabbs[i].max.z = points[3 * i + 2];
    }
    // Get the extent
    AABB* extent = (struct AABB*) malloc(sizeof(struct AABB));
    getExtent(extent, points, m_num_objects);
    
    gpuErrchk(hipMalloc(&this->m_d_extent, sizeof(struct AABB)));
    gpuErrchk(hipMemcpy(this->m_d_extent, extent, sizeof(struct AABB), hipMemcpyHostToDevice));

    AABB* d_aabbs;
    gpuErrchk(hipMalloc(&d_aabbs, sizeof(struct AABB) * num_points));
    gpuErrchk(hipMemcpy(d_aabbs, aabbs, sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    int size_morton = num_points * sizeof(unsigned long long int);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Get the morton codes of the points
    unsigned long long int* d_morton_codes;
    gpuErrchk(hipMalloc(&d_morton_codes, size_morton));

    compute_morton_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (d_aabbs, this->m_d_extent, d_morton_codes, num_points);
    
    gpuErrchk(hipPeekAtLastError());
    
    gpuErrchk( hipFree(d_aabbs) );
    
    unsigned long long int* morton_codes = (unsigned long long int*)
                    malloc(sizeof(unsigned long long int) * num_points);

    gpuErrchk( hipMemcpy(morton_codes, d_morton_codes, size_morton, hipMemcpyDeviceToHost) );
    
    gpuErrchk( hipFree(d_morton_codes) );

    // Create array of indices with an index for each point
    unsigned int* indices = (unsigned int*)
        malloc(sizeof(unsigned int) * num_points);

    for(int i = 0; i < num_points; i++)
    {
        indices[i] = i;
    }

    // Sort the indices according to the corresponding morton codes
    thrust::sort_by_key(morton_codes, morton_codes + num_points, 
                        indices);
    
    // Sort the AABBs by the indices
    AABB* sorted_aabbs = (AABB*) malloc(sizeof(AABB) * num_points);
   
    for(int i = 0; i < num_points; i++)
    {
        sorted_aabbs[i] = aabbs[indices[i]];
    }

    gpuErrchk(hipPeekAtLastError());

    // Create the nodes
    gpuErrchk(hipMalloc(&this->m_d_nodes, sizeof(struct BVHNode) * m_num_nodes));

    AABB* d_sorted_aabbs;
    gpuErrchk(hipMalloc(&d_sorted_aabbs, 
        sizeof(struct AABB) * num_points));

    gpuErrchk(hipMemcpy(d_sorted_aabbs, sorted_aabbs, 
        sizeof(struct AABB) * num_points, hipMemcpyHostToDevice));

    // Initialize the tree
    initialize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (this->m_d_nodes, d_sorted_aabbs, num_points);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipFree(d_sorted_aabbs));

    // Construct the tree
    this->m_root_node = (unsigned int*)
        malloc(sizeof(unsigned int));
    this->m_root_node[0] = UINT_MAX;

    unsigned int* d_root_node;
    gpuErrchk(hipMalloc(&d_root_node, sizeof(unsigned int)));

    gpuErrchk(hipMemcpy(d_root_node, m_root_node, sizeof(unsigned int), 
                hipMemcpyHostToDevice));

    unsigned long long int* d_sorted_morton_codes;
    gpuErrchk(hipMalloc(&d_sorted_morton_codes, size_morton));

    gpuErrchk(hipMemcpy(d_sorted_morton_codes, morton_codes, 
            size_morton, hipMemcpyHostToDevice));

    construct_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (this->m_d_nodes, d_root_node, d_sorted_morton_codes, num_points);

    // Optimize the tree
    if(this->m_leaf_size > 1)
    {
        unsigned int* valid = (unsigned int*)
            malloc(sizeof(unsigned int) * this->m_num_nodes);

        for(int i = 0; i < this->m_num_nodes; i++)
        {
            valid[i] = 1;
        }

        unsigned int* d_valid;
        hipMalloc(&d_valid, sizeof(unsigned int) * this->m_num_nodes);

        hipMemcpy(d_valid, valid, 
            sizeof(unsigned int) * this->m_num_nodes,
            hipMemcpyHostToDevice);

        optimize_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (this->m_d_nodes, d_root_node, d_valid, this->m_leaf_size, this->m_num_objects);

        hipMemcpy(valid, d_valid, 
            sizeof(unsigned int) * this->m_num_nodes,
            hipMemcpyDeviceToHost);

        // Compact tree to increase bandwidth
        if(this->m_compact)
        {
            // Get the cumulative sum of valid, but start with 0
            unsigned int* valid_sums = (unsigned int*)
                malloc(sizeof(unsigned int) * this->m_num_nodes + 1);

            valid_sums[0] = 0;
            for(int i = 1; i < this->m_num_nodes + 1; i++)
            {
                valid_sums[i] = valid_sums[i - 1] + valid[i - 1];
            }

            // Number of the actually used nodes after optimizing
            unsigned int new_node_count = valid_sums[this->m_num_nodes];

            // Calculate the isum parameter
            unsigned int* isum = (unsigned int*)
                malloc(sizeof(unsigned int) * this->m_num_nodes);

            for(int i = 0; i < this->m_num_nodes; i++)
            {
                isum[i] = i - valid_sums[i];
            }
            unsigned int free_indices_size = isum[new_node_count];

            unsigned int* free_indices;

            // Reuse valid space, since its not needed anymore
            free_indices = &valid[0];

            // Upload
            unsigned int* d_valid_sums;
            unsigned int* d_isum;
            unsigned int* d_free_indices;

            hipMalloc(&d_valid_sums, sizeof(unsigned int) * this->m_num_nodes + 1);
            hipMalloc(&d_isum, sizeof(unsigned int) * this->m_num_nodes);
            hipMalloc(&d_free_indices, sizeof(unsigned int) * free_indices_size);

            hipMemcpy(d_valid_sums, valid_sums,
                sizeof(unsigned int) * this->m_num_nodes + 1,
                hipMemcpyHostToDevice);
            hipMemcpy(d_isum, isum,
                sizeof(unsigned int) * this->m_num_nodes,
                hipMemcpyHostToDevice);
            hipMemcpy(d_free_indices, free_indices,
                sizeof(unsigned int) * free_indices_size,
                hipMemcpyHostToDevice);
            
            int threadsPerBlock = 256;
            int blocksPerGrid = (new_node_count + threadsPerBlock - 1) 
                        / threadsPerBlock;

            compute_free_indices_kernel<<<blocksPerGrid, threadsPerBlock>>>
                (d_valid_sums, d_isum, d_free_indices, new_node_count);

            hipMemcpy(valid_sums, d_valid_sums, 
                sizeof(unsigned int) * this->m_num_nodes + 1,
                hipMemcpyDeviceToHost);

            // get the sum of the first object that has to be moved
            unsigned int first_moved = valid_sums[new_node_count];

            threadsPerBlock = 256;
            blocksPerGrid = (this->m_num_nodes + threadsPerBlock - 1) 
                        / threadsPerBlock;

            // self.nodes, root_node, valid_sums_aligned, free, first_moved, new_node_count, self.num_nodes
            compact_tree_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (this->m_d_nodes, d_root_node, d_valid_sums, d_free_indices, first_moved, new_node_count, this->m_num_nodes);

            this->m_num_nodes = new_node_count;
            free(valid_sums);
            free(isum);

            gpuErrchk( hipFree(d_valid_sums) );
            gpuErrchk( hipFree(d_isum) );
            gpuErrchk( hipFree(d_free_indices) );
        }
        gpuErrchk( hipFree(d_valid) );

        free(valid);
    }
    
    gpuErrchk(hipMemcpy(m_root_node, d_root_node, 
            sizeof(unsigned int), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_root_node));
    gpuErrchk(hipFree(d_sorted_morton_codes));

    
    // Upload points to GPU
    gpuErrchk( hipMalloc(&this->m_d_points,
        sizeof(float) * 3 * num_points) );
    gpuErrchk( hipMemcpy(this->m_d_points, points,
        sizeof(float) * 3 * num_points,
        hipMemcpyHostToDevice) );
    // Upload sorted indices to GPU
    gpuErrchk( hipMalloc(&this->m_d_sorted_indices,
        sizeof(unsigned int) * num_points) );
    gpuErrchk( hipMemcpy(this->m_d_sorted_indices, indices,
        sizeof(unsigned int) * num_points,
        hipMemcpyHostToDevice) );

    free(aabbs);
    free(morton_codes);
    free(indices);
    free(sorted_aabbs);

    return;
}

#define CUDA_SAFE_CALL(x) \
    do { \
        hipError_t result = x; \
        if (result != hipSuccess) { \
            const char *msg; \
            hipDrvGetErrorName(result, &msg); \
            std::cerr << "\nerror: " #x " failed with error " \
            << msg << '\n'; \
            exit(1); \
        } \
    } while(0)

#define NVRTC_SAFE_CALL(x)                                        \
    do {                                                            \
        hiprtcResult result = x;                                       \
        if (result != HIPRTC_SUCCESS) {                                \
            std::cerr << "\nerror: " #x " failed with error "           \
                        << hiprtcGetErrorString(result) << '\n';           \
            exit(1);                                                    \
        }                                                             \
    } while(0)

void LBVHIndex::kSearch(
    float* query_points, 
    size_t num_queries,
    int K, 
    unsigned int* n_neighbors_out, 
    unsigned int* indices_out, 
    float* distances_out
) const
{   
    float radius = FLT_MAX;
    
    this->process_queries(query_points, num_queries, K, radius,
        n_neighbors_out, indices_out, distances_out);
}

void LBVHIndex::kSearch_dev_ptr(
    float* d_query_points, 
    size_t num_queries,
    int K, 
    unsigned int* d_n_neighbors_out, 
    unsigned int* d_indices_out, 
    float* d_distances_out
) const
{
    float radius = FLT_MAX;
    
    this->process_queries_dev_ptr(d_query_points, num_queries, K, radius,
        d_n_neighbors_out, d_indices_out, d_distances_out);
}

void LBVHIndex::radiusSearch(
    float* query_points, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* n_neighbors_out, 
    unsigned int* indices_out, 
    float* distances_out
) const
{   
    this->process_queries(query_points, num_queries, K, r,
        n_neighbors_out, indices_out, distances_out);
}

void LBVHIndex::radiusSearch_dev_ptr(
    float* d_query_points, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* d_n_neighbors_out, 
    unsigned int* d_indices_out, 
    float* d_distances_out
) const
{
    this->process_queries_dev_ptr(d_query_points, num_queries, K, r,
        d_n_neighbors_out, d_indices_out, d_distances_out);
}

void LBVHIndex::process_queries(
    float* queries_raw, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* n_neighbors_out, 
    unsigned int* indices_out, 
    float* distances_out
) const
{
    unsigned int* d_n_neighbors_out; 
    unsigned int* d_indices_out; 
    float* d_distances_out;
    
    // Allocate output buffer
    gpuErrchk( hipMalloc(&d_indices_out, sizeof(unsigned int) * num_queries * K) ); // Now here out of memory#########################
    gpuErrchk( hipMalloc(&d_distances_out, sizeof(float) * num_queries * K) );
    gpuErrchk( hipMalloc(&d_n_neighbors_out, sizeof(unsigned int) * num_queries) );

    // Upload
    float* d_query_points;
    gpuErrchk( hipMalloc(&d_query_points, sizeof(float) * 3 * num_queries) );
    gpuErrchk( hipMemcpy(d_query_points, queries_raw,
        sizeof(float) * 3 * num_queries,
        hipMemcpyHostToDevice) );
    
    // Compute on GPU
    this->process_queries_dev_ptr(
        d_query_points, 
        num_queries,
        K, 
        r,
        d_n_neighbors_out, 
        d_indices_out, 
        d_distances_out
    );
    
    // Download
    gpuErrchk( hipMemcpy(indices_out, d_indices_out,
            sizeof(unsigned int) * num_queries * K,
            hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(distances_out, d_distances_out,
            sizeof(float) * num_queries * K,
            hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(n_neighbors_out, d_n_neighbors_out,
            sizeof(unsigned int) * num_queries,
            hipMemcpyDeviceToHost) );


    hipFree(d_indices_out);
    hipFree(d_distances_out);
    hipFree(d_n_neighbors_out);
    hipFree(d_query_points);
}

void LBVHIndex::process_queries_dev_ptr(
    float* d_query_points, 
    size_t num_queries,
    int K, 
    float r,
    unsigned int* d_n_neighbors_out, 
    unsigned int* d_indices_out, 
    float* d_distances_out
) const
{
    // Get the Query Kernel
    std::string kernel_file = "query_knn_kernels.cu";
    std::string kernel_name = "query_knn_kernel";
    std::string kernel_dir = std::string(LBVH_KERNEL_DIR);
    std::string kernel_path = kernel_dir + "/" + kernel_file;

    // Read the kernel file
    std::ifstream in(kernel_path);
    std::string cu_src((std::istreambuf_iterator<char>(in)),
        std::istreambuf_iterator<char>());

    // Get the ptx of the kernel
    std::string ptx_src;

    getPtxFromCuString(
        ptx_src, 
        kernel_name.c_str(), 
        cu_src.c_str(), 
        K
    );
    
    // Get cuda module and function
    hipModule_t module;
    hipFunction_t kernel;
    CUDA_SAFE_CALL(hipInit(0));
    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx_src.c_str(), 0, 0, 0));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernel_name.c_str()));

    // Prepare kernel launch
    unsigned int* sorted_queries = (unsigned int*) 
                malloc(sizeof(unsigned int) * num_queries);

    for(int i = 0; i < num_queries; i++)
    {
        sorted_queries[i] = i;
    }

    // Sort the queries according to their morton codes
    if(this->m_sort_queries)
    {
        unsigned long long int* morton_codes_query =
            (unsigned long long int*)
            malloc(sizeof(unsigned long long int) * num_queries);

        unsigned long long int* d_morton_codes_query;
        hipMalloc(&d_morton_codes_query, 
            sizeof(unsigned long long int) * num_queries);

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;

        compute_morton_points_kernel<<<blocksPerGrid, threadsPerBlock>>>
            (d_query_points, this->m_d_extent, d_morton_codes_query, num_queries);

        hipMemcpy(morton_codes_query, d_morton_codes_query,
            sizeof(unsigned long long int) * num_queries,
            hipMemcpyDeviceToHost);
        
        thrust::sort_by_key(morton_codes_query, morton_codes_query + num_queries, 
                        sorted_queries);
    }

    // Upload
    unsigned int* d_sorted_queries;
    gpuErrchk( hipMalloc(&d_sorted_queries, sizeof(unsigned int) * num_queries) );
    gpuErrchk( hipMemcpy(d_sorted_queries, sorted_queries,
            sizeof(unsigned int) * num_queries,
            hipMemcpyHostToDevice) );

    // Params need to be cast to const
    BVHNode* d_nodes = const_cast<BVHNode*>(this->m_d_nodes);
    float* d_points = const_cast<float*>(this->m_d_points);
    unsigned int* d_sorted_indices = const_cast<unsigned int*>(this->m_d_sorted_indices);
    unsigned int root_node = this->m_root_node[0];
    float radius = r;

    void *params[] = 
    {
        &d_nodes,
        &d_points,
        &d_sorted_indices,
        &root_node,
        &radius,
        &d_query_points,
        &d_sorted_queries,
        &num_queries,
        &d_indices_out,
        &d_distances_out,
        &d_n_neighbors_out
    };

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;
   
    // Launch the kernel
    CUDA_SAFE_CALL( hipModuleLaunchKernel(kernel, 
        blocksPerGrid, 1, 1,  // grid dim
        threadsPerBlock, 1, 1,    // block dim
        0, NULL,    // shared mem and stream
        params,       // arguments
        0
    ) );      

    hipFree(d_query_points);
    hipFree(d_sorted_queries);

    free(sorted_queries);
    
    return;
}

 void LBVHIndex::calculate_normals(
    float* normals, 
    size_t num_normals,
    float* queries, 
    size_t num_queries,
    int K,
    const unsigned int* n_neighbors_in, 
    const unsigned int* indices_in
)   const
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_normals + threadsPerBlock - 1) 
                        / threadsPerBlock;

    // Create device memory
    float* d_queries;
    gpuErrchk( hipMalloc(&d_queries, 
        sizeof(float) * 3 * num_queries) );
    
    float* d_normals;
    gpuErrchk( hipMalloc(&d_normals, 
        sizeof(float) * 3 * num_normals) );

    unsigned int* d_n_neighbors_in;
    gpuErrchk( hipMalloc(&d_n_neighbors_in, 
        sizeof(unsigned int) * num_queries) );

    unsigned int* d_indices_in;
    gpuErrchk( hipMalloc(&d_indices_in, 
        sizeof(unsigned int) * K * num_queries) );

    // Upload
    gpuErrchk( hipMemcpy(d_queries, queries,
        sizeof(float) * 3 * num_queries,
        hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_n_neighbors_in, n_neighbors_in,
        sizeof(unsigned int) * num_queries, 
        hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_indices_in, indices_in,
        sizeof(unsigned int) * K * num_queries, 
        hipMemcpyHostToDevice) );
    
    // Call the normals kernel
    calculate_normals_kernel<<<blocksPerGrid, threadsPerBlock>>>
        (this->m_d_points, d_queries, num_queries, K, d_n_neighbors_in, d_indices_in,
        d_normals, this->m_flip_x, this->m_flip_y, this->m_flip_z);
    
    // Download the normals
    gpuErrchk( hipMemcpy(normals, d_normals,
        sizeof(float) * 3 * num_normals,
        hipMemcpyDeviceToHost) );
    
    hipFree(d_queries);
    hipFree(d_normals);
    hipFree(d_indices_in);
    hipFree(d_n_neighbors_in);
}


void LBVHIndex::knn_normals(
    int K,
    float* normals, 
    size_t num_normals
)
{
    float* d_query_points = this->m_d_points;
    unsigned int* d_sorted_queries = this->m_d_sorted_indices;
    size_t num_queries = num_normals;
    
    // Get the KNN Normals Kernel
    std::string kernel_file = "knn_normals_kernel.cu";
    std::string kernel_name = "knn_normals_kernel";
    std::string kernel_dir = std::string(LBVH_KERNEL_DIR);
    std::string kernel_path = kernel_dir + "/" + kernel_file;
    
    // Read the kernel file
    std::ifstream in(kernel_path);
    std::string cu_src((std::istreambuf_iterator<char>(in)),
        std::istreambuf_iterator<char>());

    // Get the ptx of the kernel
    std::string ptx_src;

    getPtxFromCuString(
        ptx_src, 
        kernel_name.c_str(), 
        cu_src.c_str(), 
        K
    );

    // Init cuda
    hipFree(0);
    
    // Get the cuda module and function
    hipModule_t module;
    hipFunction_t kernel;

    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx_src.c_str(), 0, 0, 0));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernel_name.c_str()));

    float* d_normals;
    gpuErrchk( hipMalloc(&d_normals, 
        sizeof(float) * 3 * num_normals) );

    float radius = FLT_MAX;

    // Gather the arguments
    void *params[] = 
    {
        &this->m_d_nodes,
        &this->m_d_points,
        &this->m_d_sorted_indices,
        &this->m_root_node[0],
        &radius,
        &d_query_points,
        &d_sorted_queries,
        &num_queries,
        &d_normals
    };

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_queries + threadsPerBlock - 1) 
                        / threadsPerBlock;
   
    // Launch the kernel
    CUDA_SAFE_CALL( hipModuleLaunchKernel(kernel, 
        blocksPerGrid, 1, 1,  // grid dim
        threadsPerBlock, 1, 1,    // block dim
        0, NULL,    // shared mem and stream
        params,       // arguments
        0
    ) );    

    gpuErrchk( hipMemcpy(normals, d_normals,
        sizeof(float) * 3 * num_normals,
        hipMemcpyDeviceToHost) );  
    
    hipFree(d_sorted_queries);
    hipFree(d_normals);
}

// Get the extent of the points 
// (minimum and maximum values in each dimension)
void LBVHIndex::getExtent(
    AABB* extent, 
    float* points, 
    size_t num_points
) const
{
    float min_x = INT_MAX;
    float min_y = INT_MAX;
    float min_z = INT_MAX;

    float max_x = INT_MIN;
    float max_y = INT_MIN;
    float max_z = INT_MIN;

    for(int i = 0; i < num_points; i++)
    {
        if(points[3 * i + 0] < min_x)
        {
            min_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] < min_y)
        {
            min_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] < min_z)
        {
            min_z = points[3 * i + 2];
        }

        if(points[3 * i + 0] > max_x)
        {
            max_x = points[3 * i + 0];
        }

        if(points[3 * i + 1] > max_y)
        {
            max_y = points[3 * i + 1];
        }

        if(points[3 * i + 2] > max_z)
        {
            max_z = points[3 * i + 2];
        }
    }
    
    extent->min.x = min_x;
    extent->min.y = min_y;
    extent->min.z = min_z;
    
    extent->max.x = max_x;
    extent->max.y = max_y;
    extent->max.z = max_z;
    
    return;
}

void LBVHIndex::getPtxFromCuString( 
    std::string& ptx, 
    const char* sample_name, 
    const char* cu_source, 
    int K
) const
{
    // Create program
    hiprtcProgram prog;
    NVRTC_SAFE_CALL( hiprtcCreateProgram( &prog, cu_source, sample_name, 0, NULL, NULL ) );

    std::string K_str = "-DK=" + std::to_string(K); 

    // Gather NVRTC options
    std::string kernel_includes = std::string("-I") + 
        std::string(LBVH_KERNEL_INCLUDES);

    std::string cuda_include = std::string("-I") + 
        std::string(CUDA_INCLUDE_DIRS);
    
    std::vector<const char*> options = {
        kernel_includes.c_str(),
        cuda_include.c_str(),
        "-std=c++17",
        K_str.c_str()
    };

    const std::string base_dir = std::string(LBVH_KERNEL_DIR);

    // JIT compile CU to PTX
    const hiprtcResult compileRes = hiprtcCompileProgram( prog, (int)options.size(), options.data() );
    
    // Retrieve log output
    size_t log_size = 0;
    NVRTC_SAFE_CALL( hiprtcGetProgramLogSize( prog, &log_size ) );

    char* log = new char[log_size];
    if( log_size > 1 )
    {
        NVRTC_SAFE_CALL( hiprtcGetProgramLog( prog, log ) );
        std::cout << log << std::endl;
    }
    
    if( compileRes != HIPRTC_SUCCESS )
        throw std::runtime_error( "NVRTC Compilation failed.\n");

    // Retrieve PTX code
    size_t ptx_size = 0;
    NVRTC_SAFE_CALL( hiprtcGetCodeSize( prog, &ptx_size ) );
    ptx.resize( ptx_size );
    NVRTC_SAFE_CALL( hiprtcGetCode( prog, &ptx[0] ) );

    // Cleanup
    NVRTC_SAFE_CALL( hiprtcDestroyProgram( &prog ) );
}

} // namespace lvr2
