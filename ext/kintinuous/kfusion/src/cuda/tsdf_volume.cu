#include "hip/hip_runtime.h"
#include <kfusion/cuda/device.hpp>
#include <kfusion/cuda/texture_binder.hpp>
#include <kfusion/tsdf_buffer.h>


using namespace kfusion::device;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume initialization

namespace kfusion
{
    namespace device
    {
		__kf_device__  void
		shift_tsdf_pointer(ushort2 ** value, kfusion::tsdf_buffer buffer)
		{
		  ///Shift the pointer by (@origin - @start)
		  *value += (buffer.tsdf_rolling_buff_origin - buffer.tsdf_memory_start);

		  ///If we land outside of the memory, make sure to "modulo" the new value
		  if(*value > buffer.tsdf_memory_end)
		  {
			*value -= (buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1); /// correction of bug found my qianyizh
		  }
	    }

        __global__ void clear_volume_kernel(TsdfVolume tsdf)
        {
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;

            if (x < tsdf.dims.x && y < tsdf.dims.y)
            {
                ushort2 *beg = tsdf.beg(x, y);
                ushort2 *end = beg + tsdf.dims.x * tsdf.dims.y * tsdf.dims.z;

                for(ushort2* pos = beg; pos != end; pos = tsdf.zstep(pos))
                    *pos = pack_tsdf (0.f, 0);
            }
        }

      __global__ void
      clearSliceKernel (TsdfVolume tsdf, const kfusion::tsdf_buffer buffer, int3 minBounds, int3 maxBounds)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		//compute relative indices
		int idX, idY;
		if(x <= minBounds.x)
			idX = x + buffer.voxels_size.x;
		else
			idX = x;
		if(y <= minBounds.y)
			idY = y + buffer.voxels_size.y;
		else
			idY = y;

        if ( x < buffer.voxels_size.x && y < buffer.voxels_size.y)
        {
            if( (idX >= minBounds.x && idX <= maxBounds.x) || (idY >= minBounds.y && idY <= maxBounds.y) )
            {
                /// BLACK ZONE => clear on all Z values
                ushort2 *beg = tsdf.beg(x, y);
                ushort2 *end = beg + tsdf.dims.x * tsdf.dims.y * tsdf.dims.z;

                for(ushort2* pos = beg; pos != end; pos = tsdf.zstep(pos))
                {
                    *pos = pack_tsdf (0.f, 0);
                }
            }
            else /* if( idX > maxBounds.x && idY > maxBounds.y) */
            {
                ///RED ZONE  => clear only appropriate Z

                ///Pointer to the first x,y,0
                ushort2 *pos = tsdf.beg(x, y);

                ///Get the size of the whole TSDF memory
                int size = buffer.tsdf_memory_end - buffer.tsdf_memory_start;

                ///Move pointer to the Z origin
                pos = tsdf.multzstep(minBounds.z, pos);

				if(maxBounds.z < 0)
				{
					pos = tsdf.multzstep(maxBounds.z, pos);
				}
                ///We make sure that we are not already before the start of the memory
                if(pos < buffer.tsdf_memory_start)
                    pos = pos + size;

                int nbSteps = abs(maxBounds.z);

				#pragma unroll
                for(int z = 0; z < nbSteps; ++z, pos = tsdf.zstep(pos))
                {
                  ///If we went outside of the memory, make sure we go back to the begining of it
                  if(pos > buffer.tsdf_memory_end)
                    pos = pos - size;

                  if (pos >= buffer.tsdf_memory_start && pos <= buffer.tsdf_memory_end) // quickfix for http://dev.pointclouds.org/issues/894
                    *pos = pack_tsdf (0.f, 0);
                }
            } //else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
        } // if ( x < VOLUME_X && y < VOLUME_Y)
      } // clearSliceKernel
    }
}

void kfusion::device::clear_volume(TsdfVolume volume)
{
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = divUp (volume.dims.x, block.x);
    grid.y = divUp (volume.dims.y, block.y);

    clear_volume_kernel<<<grid, block>>>(volume);
    cudaSafeCall ( hipGetLastError () );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume integration

namespace kfusion
{
    namespace device
    {
        texture<float, 2> dists_tex(0, hipFilterModePoint, hipAddressModeBorder, cudaCreateChannelDescHalf());

        struct TsdfIntegrator
        {
            Aff3f vol2cam;
            Projector proj;
            int2 dists_size;

            float tranc_dist_inv;

            __kf_device__
            void operator()(TsdfVolume& volume, tsdf_buffer& buffer) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume.dims.x || y >= volume.dims.y)
                    return;

                //float3 zstep = vol2cam.R * make_float3(0.f, 0.f, volume.voxel_size.z);
                float3 zstep = make_float3(vol2cam.R.data[0].z, vol2cam.R.data[1].z, vol2cam.R.data[2].z) * volume.voxel_size.z;

                float3 vx = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0);
                float3 vc = vol2cam * vx; //tranform from volume coo frame to camera one

                TsdfVolume::elem_type* vptr = volume.beg(x, y);
                for(int i = 0; i < volume.dims.z; ++i, vc += zstep, vptr = volume.zstep(vptr))
                {
                    float2 coo = proj(vc);

                    //#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
                    // this is actually workaround for kepler. it doesn't return 0.f for texture
                    // fetches for out-of-border coordinates even for cudaaddressmodeborder mode
                    if (coo.x < 0 || coo.y < 0 || coo.x >= dists_size.x || coo.y >= dists_size.y)
                        continue;
                    //#endif
                    float Dp = tex2D(dists_tex, coo.x, coo.y);
                    if(Dp == 0 || vc.z <= 0)
                        continue;

                    float sdf = Dp - __fsqrt_rn(dot(vc, vc)); //Dp - norm(v)

                    if (sdf >= -volume.trunc_dist)
                    {
                        float tsdf = fmin(1.f, sdf * tranc_dist_inv);

                        //read and unpack
                        int weight_prev;

           				ushort2* pos = const_cast<ushort2*> (vptr);

						shift_tsdf_pointer (&pos, buffer);

                        float tsdf_prev = unpack_tsdf (gmem::LdCs(pos), weight_prev);

                        float tsdf_new = __fdividef(__fmaf_rn(tsdf_prev, weight_prev, tsdf), weight_prev + 1);
                        int weight_new = min (weight_prev + 1, volume.max_weight);

                        //pack and write
                        gmem::StCs(pack_tsdf (tsdf_new, weight_new), pos);
                    }
                }  // for(;;)
            }
        };

        __global__ void integrate_kernel( const TsdfIntegrator integrator, TsdfVolume volume, tsdf_buffer buffer) { integrator(volume, buffer); };
    }
}

void kfusion::device::integrate(const PtrStepSz<ushort>& dists, TsdfVolume& volume, tsdf_buffer& buffer, const Aff3f& aff, const Projector& proj)
{
    TsdfIntegrator ti;
    ti.dists_size = make_int2(dists.cols, dists.rows);
    ti.vol2cam = aff;
    ti.proj = proj;
    ti.tranc_dist_inv = 1.f/volume.trunc_dist;

    dists_tex.filterMode = hipFilterModePoint;
    dists_tex.addressMode[0] = hipAddressModeBorder;
    dists_tex.addressMode[1] = hipAddressModeBorder;
    dists_tex.addressMode[2] = hipAddressModeBorder;
    TextureBinder binder(dists, dists_tex, cudaCreateChannelDescHalf()); (void)binder;

    dim3 block(32, 8);
    dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y));

    integrate_kernel<<<grid, block>>>(ti, volume, buffer);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize() );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume ray casting

namespace kfusion
{
    namespace device
    {
        __kf_device__ void intersect(float3 ray_org, float3 ray_dir, /*float3 box_min,*/ float3 box_max, float &tnear, float &tfar)
        {
            const float3 box_min = make_float3(0.f, 0.f, 0.f);

            // compute intersection of ray with all six bbox planes
            float3 invR = make_float3(1.f/ray_dir.x, 1.f/ray_dir.y, 1.f/ray_dir.z);
            float3 tbot = invR * (box_min - ray_org);
            float3 ttop = invR * (box_max - ray_org);

            // re-order intersections to find smallest and largest on each axis
            float3 tmin = make_float3(fminf(ttop.x, tbot.x), fminf(ttop.y, tbot.y), fminf(ttop.z, tbot.z));
            float3 tmax = make_float3(fmaxf(ttop.x, tbot.x), fmaxf(ttop.y, tbot.y), fmaxf(ttop.z, tbot.z));

            // find the largest tmin and the smallest tmax
            tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
            tfar  = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));
        }

        template<typename Vol>
        __kf_device__ float interpolate(const Vol& volume,const tsdf_buffer& buffer, const float3& p_voxels)
        {
            float3 cf = p_voxels;

            //rounding to negative infinity
            int3 g = make_int3(__float2int_rd (cf.x), __float2int_rd (cf.y), __float2int_rd (cf.z));

            if (g.x < 0 || g.x >= volume.dims.x - 1 || g.y < 0 || g.y >= volume.dims.y - 1 || g.z < 0 || g.z >= volume.dims.z - 1)
                return numeric_limits<float>::quiet_NaN();

            float a = cf.x - g.x;
            float b = cf.y - g.y;
            float c = cf.z - g.z;

            float tsdf = 0.f;
            ushort2* pos1 = const_cast<ushort2*> (volume(g.x + 0, g.y + 0, g.z + 0));
            ushort2* pos2 = const_cast<ushort2*> (volume(g.x + 0, g.y + 0, g.z + 1));
            ushort2* pos3 = const_cast<ushort2*> (volume(g.x + 0, g.y + 1, g.z + 0));
            ushort2* pos4 = const_cast<ushort2*> (volume(g.x + 0, g.y + 1, g.z + 1));
            ushort2* pos5 = const_cast<ushort2*> (volume(g.x + 1, g.y + 0, g.z + 0));
            ushort2* pos6 = const_cast<ushort2*> (volume(g.x + 1, g.y + 0, g.z + 1));
            ushort2* pos7 = const_cast<ushort2*> (volume(g.x + 1, g.y + 1, g.z + 0));
            ushort2* pos8 = const_cast<ushort2*> (volume(g.x + 1, g.y + 1, g.z + 1));

			shift_tsdf_pointer (&pos1, buffer);
			shift_tsdf_pointer (&pos2, buffer);
			shift_tsdf_pointer (&pos3, buffer);
			shift_tsdf_pointer (&pos4, buffer);
			shift_tsdf_pointer (&pos5, buffer);
			shift_tsdf_pointer (&pos6, buffer);
			shift_tsdf_pointer (&pos7, buffer);
			shift_tsdf_pointer (&pos8, buffer);

            tsdf += unpack_tsdf(*pos1) * (1 - a) * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*pos2) * (1 - a) * (1 - b) *      c;
            tsdf += unpack_tsdf(*pos3) * (1 - a) *      b  * (1 - c);
            tsdf += unpack_tsdf(*pos4) * (1 - a) *      b  *      c;
            tsdf += unpack_tsdf(*pos5) *      a  * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*pos6) *      a  * (1 - b) *      c;
            tsdf += unpack_tsdf(*pos7) *      a  *      b  * (1 - c);
            tsdf += unpack_tsdf(*pos8) *      a  *      b  *      c;
            return tsdf;
        }

        struct TsdfRaycaster
        {
            TsdfVolume volume;
            tsdf_buffer buffer;

            Aff3f aff;
            Mat3f Rinv;

            Vec3f volume_size;
            Reprojector reproj;
            float time_step;
            float3 gradient_delta;
            float3 voxel_size_inv;

            TsdfRaycaster(const TsdfVolume& volume, tsdf_buffer& buffer, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& _reproj);

            __kf_device__
            float fetch_tsdf(const float3& p) const
            {
                //rounding to nearest even
                int x = __float2int_rn (p.x * voxel_size_inv.x);
                int y = __float2int_rn (p.y * voxel_size_inv.y);
                int z = __float2int_rn (p.z * voxel_size_inv.z);
				ushort2* pos = const_cast<ushort2*> (volume(x,y,z));
				shift_tsdf_pointer (&pos, buffer);
                return unpack_tsdf(*pos);
            }

            __kf_device__
            void operator()(PtrStepSz<ushort> depth, PtrStep<Normal> normals) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= depth.cols || y >= depth.rows)
                    return;

                const float qnan = numeric_limits<float>::quiet_NaN();

                depth(y, x) = 0;
                normals(y, x) = make_float4(qnan, qnan, qnan, qnan);

                float3 ray_org = aff.t;
                float3 ray_dir = normalized( aff.R * reproj(x, y, 1.f) );

                // We do subtract voxel size to minimize checks after
                // Note: origin of volume coordinate is placeed
                // in the center of voxel (0,0,0), not in the corener of the voxel!
                float3 box_max = volume_size - volume.voxel_size;

                float tmin, tmax;
                intersect(ray_org, ray_dir, box_max, tmin, tmax);

                const float min_dist = 0.f;
                tmin = fmax(min_dist, tmin);
                if (tmin >= tmax)
                    return;

                tmax -= time_step;
                float3 vstep = ray_dir * time_step;
                float3 next = ray_org + ray_dir * tmin;

                float tsdf_next = fetch_tsdf(next);
                for (float tcurr = tmin; tcurr < tmax; tcurr += time_step)
                {
                    float tsdf_curr = tsdf_next;
                    float3     curr = next;
                    next += vstep;

                    tsdf_next = fetch_tsdf(next);
                    if (tsdf_curr < 0.f && tsdf_next > 0.f)
                        break;

                    if (tsdf_curr > 0.f && tsdf_next < 0.f)
                    {
                        float Ft   = interpolate(volume, buffer, curr * voxel_size_inv);
                        float Ftdt = interpolate(volume, buffer, next * voxel_size_inv);

                        float Ts = tcurr - __fdividef(time_step * Ft, Ftdt - Ft);

                        float3 vertex = ray_org + ray_dir * Ts;
                        float3 normal = compute_normal(vertex);

                        if (!isnan(normal.x * normal.y * normal.z))
                        {
                            normal = Rinv * normal;
                            vertex = Rinv * (vertex - aff.t);

                            normals(y, x) = make_float4(normal.x, normal.y, normal.z, 0);
                            depth(y, x) = static_cast<ushort>(vertex.z * 1000);
                        }
                        break;
                    }
                } /* for (;;) */
            }

            __kf_device__
            void operator()(PtrStepSz<Point> points, PtrStep<Normal> normals) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= points.cols || y >= points.rows)
                    return;

                const float qnan = numeric_limits<float>::quiet_NaN();

                points(y, x) = normals(y, x) = make_float4(qnan, qnan, qnan, qnan);

                float3 ray_org = aff.t;
                float3 ray_dir = normalized( aff.R * reproj(x, y, 1.f) );

                // We do subtract voxel size to minimize checks after
                // Note: origin of volume coordinate is placeed
                // in the center of voxel (0,0,0), not in the corener of the voxel!
                float3 box_max = volume_size - volume.voxel_size;

                float tmin, tmax;
                intersect(ray_org, ray_dir, box_max, tmin, tmax);

                const float min_dist = 0.f;
                tmin = fmax(min_dist, tmin);
                if (tmin >= tmax)
                    return;

                tmax -= time_step;
                float3 vstep = ray_dir * time_step;
                float3 next = ray_org + ray_dir * tmin;

                float tsdf_next = fetch_tsdf(next);
                for (float tcurr = tmin; tcurr < tmax; tcurr += time_step)
                {
                    float tsdf_curr = tsdf_next;
                    float3     curr = next;
                    next += vstep;

                    tsdf_next = fetch_tsdf(next);
                    if (tsdf_curr < 0.f && tsdf_next > 0.f)
                        break;

                    if (tsdf_curr > 0.f && tsdf_next < 0.f)
                    {
                        float Ft   = interpolate(volume, buffer, curr * voxel_size_inv);
                        float Ftdt = interpolate(volume, buffer, next * voxel_size_inv);

                        float Ts = tcurr - __fdividef(time_step * Ft, Ftdt - Ft);

                        float3 vertex = ray_org + ray_dir * Ts;
                        float3 normal = compute_normal(vertex);

                        if (!isnan(normal.x * normal.y * normal.z))
                        {
                            normal = Rinv * normal;
                            vertex = Rinv * (vertex - aff.t);

                            normals(y, x) = make_float4(normal.x, normal.y, normal.z, 0.f);
                            points(y, x) = make_float4(vertex.x, vertex.y, vertex.z, 0.f);
                        }
                        break;
                    }
                } /* for (;;) */
            }


            __kf_device__
            float3 compute_normal(const float3& p) const
            {
                float3 n;

                float Fx1 = interpolate(volume, buffer, make_float3(p.x + gradient_delta.x, p.y, p.z) * voxel_size_inv);
                float Fx2 = interpolate(volume, buffer, make_float3(p.x - gradient_delta.x, p.y, p.z) * voxel_size_inv);
                n.x = __fdividef(Fx1 - Fx2, gradient_delta.x);

                float Fy1 = interpolate(volume, buffer, make_float3(p.x, p.y + gradient_delta.y, p.z) * voxel_size_inv);
                float Fy2 = interpolate(volume, buffer, make_float3(p.x, p.y - gradient_delta.y, p.z) * voxel_size_inv);
                n.y = __fdividef(Fy1 - Fy2, gradient_delta.y);

                float Fz1 = interpolate(volume, buffer, make_float3(p.x, p.y, p.z + gradient_delta.z) * voxel_size_inv);
                float Fz2 = interpolate(volume, buffer, make_float3(p.x, p.y, p.z - gradient_delta.z) * voxel_size_inv);
                n.z = __fdividef(Fz1 - Fz2, gradient_delta.z);

                return normalized (n);
            }
        };

        inline TsdfRaycaster::TsdfRaycaster(const TsdfVolume& _volume, tsdf_buffer& _buffer, const Aff3f& _aff, const Mat3f& _Rinv, const Reprojector& _reproj)
            : volume(_volume), buffer(_buffer), aff(_aff), Rinv(_Rinv), reproj(_reproj) {}

        __global__ void raycast_kernel(const TsdfRaycaster raycaster, PtrStepSz<ushort> depth, PtrStep<Normal> normals)
        { raycaster(depth, normals); };

        __global__ void raycast_kernel(const TsdfRaycaster raycaster, PtrStepSz<Point> points, PtrStep<Normal> normals)
        { raycaster(points, normals); };

    }
}

void kfusion::device::raycast(const TsdfVolume& volume, tsdf_buffer& buffer, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& reproj,
                              Depth& depth, Normals& normals, float raycaster_step_factor, float gradient_delta_factor)
{
    TsdfRaycaster rc(volume, buffer, aff, Rinv, reproj);

    rc.volume_size = volume.voxel_size * volume.dims;
    rc.time_step = volume.trunc_dist * raycaster_step_factor;
    rc.gradient_delta = volume.voxel_size * gradient_delta_factor;
    rc.voxel_size_inv = 1.f/volume.voxel_size;

    dim3 block(32, 8);
    dim3 grid (divUp (depth.cols(), block.x), divUp (depth.rows(), block.y));

    raycast_kernel<<<grid, block>>>(rc, (PtrStepSz<ushort>)depth, normals);
    cudaSafeCall (hipGetLastError ());
}


void kfusion::device::raycast(const TsdfVolume& volume, tsdf_buffer& buffer, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& reproj,
                              Points& points, Normals& normals, float raycaster_step_factor, float gradient_delta_factor)
{
    TsdfRaycaster rc(volume, buffer, aff, Rinv, reproj);

    rc.volume_size = volume.voxel_size * volume.dims;
    rc.time_step = volume.trunc_dist * raycaster_step_factor;
    rc.gradient_delta = volume.voxel_size * gradient_delta_factor;
    rc.voxel_size_inv = 1.f/volume.voxel_size;

    dim3 block(32, 8);
    dim3 grid (divUp (points.cols(), block.x), divUp (points.rows(), block.y));

    raycast_kernel<<<grid, block>>>(rc, (PtrStepSz<Point>)points, normals);
    cudaSafeCall (hipGetLastError ());
}

////////////////////////////////////////////////////////////////////////////////////////
/// Volume cloud exctraction

namespace kfusion
{
    namespace device
    {

        ////////////////////////////////////////////////////////////////////////////////////////
        ///// Prefix Scan utility

        enum ScanKind { exclusive, inclusive };

        template<ScanKind Kind, class T>
        __kf_device__ T scan_warp ( volatile T *ptr, const unsigned int idx = threadIdx.x )
        {
            const unsigned int lane = idx & 31;       // index of thread in warp (0..31)

            if (lane >=  1) ptr[idx] = ptr[idx -  1] + ptr[idx];
            if (lane >=  2) ptr[idx] = ptr[idx -  2] + ptr[idx];
            if (lane >=  4) ptr[idx] = ptr[idx -  4] + ptr[idx];
            if (lane >=  8) ptr[idx] = ptr[idx -  8] + ptr[idx];
            if (lane >= 16) ptr[idx] = ptr[idx - 16] + ptr[idx];

            if (Kind == inclusive)
                return ptr[idx];
            else
                return (lane > 0) ? ptr[idx - 1] : 0;
        }


        __device__ int global_count = 0;
        __device__ int output_count;
        __device__ unsigned int blocks_done = 0;


        struct FullScan6
        {
            enum
            {
                CTA_SIZE_X = 32,
                CTA_SIZE_Y = 6,
                CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,

                MAX_LOCAL_POINTS = 3
            };

            TsdfVolume volume;
            Aff3f aff;

            FullScan6(const TsdfVolume& vol) : volume(vol) {}

            /*__kf_device__ float fetch(int x, int y, int z, int& weight) const
            {
                return unpack_tsdf(*volume(x, y, z), weight);
            }*/

            __kf_device__ float fetch (const kfusion::tsdf_buffer& buffer, int x, int y, int z, int& weight) const
			{
			  const ushort2* tmp_pos = volume(x, y, z);
			  ushort2* pos = const_cast<ushort2*> (tmp_pos);

			  shift_tsdf_pointer (&pos, buffer);

			  return unpack_tsdf (*pos, weight);
			}

            __kf_device__ void operator () (PtrSz<Point> output, const tsdf_buffer buffer) const
            {
                int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
                int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;
#if __CUDA_ARCH__ < 200
                __shared__ int cta_buffer[CTA_SIZE];
#endif

#if __CUDA_ARCH__ >= 120
                if (__all (x >= volume.dims.x) || __all (y >= volume.dims.y))
                    return;
#else
                if (Emulation::All(x >= volume.dims.x, cta_buffer) || Emulation::All(y >= volume.dims.y, cta_buffer))
                    return;
#endif

                float3 V;
                V.x = (x + 0.5f) * volume.voxel_size.x;
                V.y = (y + 0.5f) * volume.voxel_size.y;

                int ftid = Block::flattenedThreadId ();

                for (int z = 0; z < volume.dims.z - 1; ++z)
                {
                    float3 points[MAX_LOCAL_POINTS];
                    int local_count = 0;

                    if (x < volume.dims.x && y < volume.dims.y)
                    {
                        int W;
                        float F = fetch(buffer, x, y, z, W);

                        if (W != 0 && F != 1.f)
                        {
                            V.z = (z + 0.5f) * volume.voxel_size.z;

                            //process dx
                            if (x + 1 < volume.dims.x)
                            {
                                int Wn;
                                float Fn = fetch(buffer, x + 1, y, z, Wn);

                                if (Wn != 0 && Fn != 1.f)
                                    if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                    {
                                        float3 p;
                                        p.y = V.y;
                                        p.z = V.z;

                                        float Vnx = V.x + volume.voxel_size.x;

                                        float d_inv = 1.f / (fabs (F) + fabs (Fn));
                                        p.x = (V.x * fabs (Fn) + Vnx * fabs (F)) * d_inv;

                                        points[local_count++] = aff * p;
                                    }
                            }  /* if (x + 1 < volume.dims.x) */

                            //process dy
                            if (y + 1 < volume.dims.y)
                            {
                                int Wn;
                                float Fn = fetch (buffer, x, y + 1, z, Wn);

                                if (Wn != 0 && Fn != 1.f)
                                    if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                    {
                                        float3 p;
                                        p.x = V.x;
                                        p.z = V.z;

                                        float Vny = V.y + volume.voxel_size.y;

                                        float d_inv = 1.f / (fabs (F) + fabs (Fn));
                                        p.y = (V.y * fabs (Fn) + Vny * fabs (F)) * d_inv;

                                        points[local_count++] = aff * p;
                                    }
                            } /*  if (y + 1 < volume.dims.y) */

                            //process dz
                            //if (z + 1 < volume.dims.z) // guaranteed by loop
                            {
                                int Wn;
                                float Fn = fetch (buffer, x, y, z + 1, Wn);

                                if (Wn != 0 && Fn != 1.f)
                                    if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                    {
                                        float3 p;
                                        p.x = V.x;
                                        p.y = V.y;

                                        float Vnz = V.z + volume.voxel_size.z;

                                        float d_inv = 1.f / (fabs (F) + fabs (Fn));
                                        p.z = (V.z * fabs (Fn) + Vnz * fabs (F)) * d_inv;

                                        points[local_count++] = aff * p;
                                    }
                            } /* if (z + 1 < volume.dims.z) */
                        } /* if (W != 0 && F != 1.f) */
                    } /* if (x < volume.dims.x && y < volume.dims.y) */

#if __CUDA_ARCH__ >= 200
                    ///not we fulfilled points array at current iteration
                    int total_warp = __popc (__ballot (local_count > 0)) + __popc (__ballot (local_count > 1)) + __popc (__ballot (local_count > 2));
#else
                    int tid = Block::flattenedThreadId();
                    cta_buffer[tid] = local_count;
                    int total_warp = Emulation::warp_reduce(cta_buffer, tid);
#endif
                    __shared__ float storage_X[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_Y[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_Z[CTA_SIZE * MAX_LOCAL_POINTS];

                    if (total_warp > 0)
                    {
                        int lane = Warp::laneId ();
                        int storage_index = (ftid >> Warp::LOG_WARP_SIZE) * Warp::WARP_SIZE * MAX_LOCAL_POINTS;

                        volatile int* cta_buffer = (int*)(storage_X + storage_index);

                        cta_buffer[lane] = local_count;
                        int offset = scan_warp<exclusive>(cta_buffer, lane);

                        if (lane == 0)
                        {
                            int old_global_count = atomicAdd (&global_count, total_warp);
                            cta_buffer[0] = old_global_count;
                        }
                        int old_global_count = cta_buffer[0];

                        for (int l = 0; l < local_count; ++l)
                        {
                            storage_X[storage_index + offset + l] = points[l].x;
                            storage_Y[storage_index + offset + l] = points[l].y;
                            storage_Z[storage_index + offset + l] = points[l].z;
                        }

                        Point *pos = output.data + old_global_count + lane;
                        for (int idx = lane; idx < total_warp; idx += Warp::STRIDE, pos += Warp::STRIDE)
                        {
                            float x = storage_X[storage_index + idx];
                            float y = storage_Y[storage_index + idx];
                            float z = storage_Z[storage_index + idx];
                            *pos = make_float4(x, y, z, 0.f);
                        }

                        bool full = (old_global_count + total_warp) >= output.size;

                        if (full)
                            break;
                    }

                } /* for(int z = 0; z < volume.dims.z - 1; ++z) */


                ///////////////////////////
                // prepare for future scans
                if (ftid == 0)
                {
                    unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
                    unsigned int value = atomicInc (&blocks_done, total_blocks);

                    //last block
                    if (value == total_blocks - 1)
                    {
                        output_count = min ((int)output.size, global_count);
                        blocks_done = 0;
                        global_count = 0;
                    }
                }
            }

			// OPERATOR USED BY EXTRACT_SLICE_AS_CLOUD.
			// This operator extracts the cloud as TSDF values and X,Y,Z indices.
			// The previous operator generates a regular point cloud in meters.
			// This one generates a TSDF Point Cloud in grid indices.
			//__kf_device__ void operator () (PtrSz<Point> output) const
			// TODO fix operator for slice download
			__kf_device__ void
			operator () (kfusion::tsdf_buffer buffer, PtrSz<Point> output, int3 minBounds, int3 maxBounds, int3 bufferShifts) const
			{
				int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
				int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

#if __CUDA_ARCH__ < 200
				__shared__ int cta_buffer[CTA_SIZE];
#endif

#if __CUDA_ARCH__ >= 120
				if (__all (x >= volume.dims.x) || __all (y >= volume.dims.y))
				return;
#else
				if (Emulation::All(x >= volume.dims.x, cta_buffer) || Emulation::All(y >= volume.dims.y, cta_buffer))
				return;
#endif
				int ftid = Block::flattenedThreadId ();

				for (int z = 0; z < volume.dims.z; ++z)
				{
					// The black zone is the name given to the subvolume within the TSDF Volume grid that is shifted out.
					// In other words, the set of points in the TSDF grid that we want to extract in order to add it to the world model being built in CPU.
					bool in_black_zone = ( (x >= minBounds.x && x <= maxBounds.x) || (y >= minBounds.y && y <= maxBounds.y) || ( z >= minBounds.z && z <= maxBounds.z) ) ;
					//bool in_black_zone = ( (x >= minBounds.x && x <= maxBounds.x) && (y >= minBounds.y && y <= maxBounds.y) && ( z >= minBounds.z && z <= maxBounds.z) ) ;
					float4 points[MAX_LOCAL_POINTS];
					int local_count = 0;
					if (x < volume.dims.x && y < volume.dims.y && in_black_zone)
					{
						int W;
						float F = fetch (buffer, x, y, z, W);
						if (W != 0 && F != 1.f && F < 0.98 && F != 0.0f && F > -0.98)
						//if (W != 0 && F != 1.f && F < 0.8 && F != 0.0f && F > -0.8)
						{
							float4 pt;
							pt.x = x + bufferShifts.x;
						    pt.y = y + bufferShifts.y;//abs((y + bufferShifts.y) - 511);
							pt.z = z + bufferShifts.z;
							pt.w = F;
							points[local_count++] = pt;
						}
					}/* if (x < VOLUME_X && y < VOLUME_Y) */

#if __CUDA_ARCH__ >= 200
                    ///not we fulfilled points array at current iteration
                    int total_warp = __popc (__ballot (local_count > 0)) + __popc (__ballot (local_count > 1)) + __popc (__ballot (local_count > 2));
#else
                    int tid = Block::flattenedThreadId();
                    cta_buffer[tid] = local_count;
                    int total_warp = Emulation::warp_reduce(cta_buffer, tid);
#endif
					__shared__ float storage_X[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_Y[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_Z[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_I[CTA_SIZE * MAX_LOCAL_POINTS];

					// local_count counts the number of zero crossing for the current thread. Now we need to merge this knowledge with the other threads
					// not we fulfilled points array at current iteration
					if (total_warp > 0) ///more than 0 zero-crossings
					{
						int lane = Warp::laneId (); ///index of thread within warp [0-31]
						int storage_index = (ftid >> Warp::LOG_WARP_SIZE) * Warp::WARP_SIZE * MAX_LOCAL_POINTS;
						// Pointer to the beginning of the current warp buffer
						volatile int* cta_buffer = (int*)(storage_X + storage_index);
						// Compute offset of current warp
						// Call in place scanning (see http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html)
						cta_buffer[lane] = local_count;
						int offset = scan_warp<exclusive>(cta_buffer, lane); //How many crossings did we have before index "lane" ?
						// We want to do only 1 operation per warp (not thread) -> because it is faster
						if (lane == 0)
						{
							int old_global_count = atomicAdd (&global_count, total_warp); ///We use atomicAdd, so that threads do not collide
							cta_buffer[0] = old_global_count;
						}
						int old_global_count = cta_buffer[0];
						// Perform compaction (dump all current crossings)
						for (int l = 0; l < local_count; ++l)
						{
							storage_X[storage_index + offset + l] = points[l].x;// x coordinates of the points we found in STORAGE_X
							storage_Y[storage_index + offset + l] = points[l].y;// y coordinates of the points we found in STORAGE_Y
							storage_Z[storage_index + offset + l] = points[l].z;// z coordinates of the points we found in STORAGE_Z
							storage_I[storage_index + offset + l] = points[l].w;// Intensity values of the points we found in STORAGE_I
						}
						// Retrieve Zero-crossings as 3D points
						//int offset_storage = old_global_count + lane;
						Point *pos = output.data + old_global_count + lane;
						for (int idx = lane; idx < total_warp; idx += Warp::STRIDE, pos += Warp::STRIDE)
						{
							float x = storage_X[storage_index + idx];
							float y = storage_Y[storage_index + idx];
							float z = storage_Z[storage_index + idx];
							float i = storage_I[storage_index + idx];
							//store_point_intensity (x, y, z, i, output.data, output_intensity.data, offset_storage);
							*pos = make_float4(x, y, z, i);
						}
						// Sanity check to make sure our output_xyz buffer is not full already
						bool full = (old_global_count + total_warp) >= output.size;
						if (full)
							break;
					}
				} /* for(int z = 0; z < VOLUME_Z - 1; ++z) */
				///////////////////////////
				// Prepare for future scans
				if (ftid == 0)
				{
					unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
					unsigned int value = atomicInc (&blocks_done, total_blocks);
					// Last block
					if (value == total_blocks - 1)
					{
						output_count = min ((int)output.size, global_count);
						blocks_done = 0;
						global_count = 0;
					}
				}
			} /* operator() */
        };

		__global__ void
		extractSliceKernel (const FullScan6 fs, PtrSz<Point> output, kfusion::tsdf_buffer buffer, int3 minBounds, int3 maxBounds, int3 bufferShifts)
		{
			fs (buffer, output, minBounds, maxBounds, bufferShifts);
		}

        __global__ void extract_kernel(const FullScan6 fs, const tsdf_buffer buffer, PtrSz<Point> output) { fs(output, buffer); }



        struct ExtractNormals
        {
            typedef float8 float8;

            TsdfVolume volume;
            tsdf_buffer buffer;
            PtrSz<Point> points;
            float3 voxel_size_inv;
            float3 gradient_delta;
            Aff3f aff;
            Mat3f Rinv;

            ExtractNormals(const TsdfVolume& vol, const tsdf_buffer& buff) : volume(vol), buffer(buff)
            {
                voxel_size_inv.x = 1.f/volume.voxel_size.x;
                voxel_size_inv.y = 1.f/volume.voxel_size.y;
                voxel_size_inv.z = 1.f/volume.voxel_size.z;
            }

            __kf_device__ int3 getVoxel (const float3& p) const
            {
                //rounding to nearest even
                int x = __float2int_rn (p.x * voxel_size_inv.x);
                int y = __float2int_rn (p.y * voxel_size_inv.y);
                int z = __float2int_rn (p.z * voxel_size_inv.z);
                return make_int3 (x, y, z);
            }

            __kf_device__ void operator () (float4* output) const
            {
                int idx = threadIdx.x + blockIdx.x * blockDim.x;

                if (idx >= points.size)
                    return;

                const float qnan = numeric_limits<float>::quiet_NaN ();
                float3 n = make_float3 (qnan, qnan, qnan);

                float3 point = Rinv * (tr(points.data[idx]) - aff.t);
                int3 g = getVoxel (point);

                if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < volume.dims.x - 2 && g.y < volume.dims.y - 2 && g.z < volume.dims.z - 2)
                {
                    float3 t;

                    t = point;
                    t.x += gradient_delta.x;;
                    float Fx1 = interpolate(volume, buffer, t * voxel_size_inv);

                    t = point;
                    t.x -= gradient_delta.x;
                    float Fx2 = interpolate(volume, buffer, t * voxel_size_inv);

                    n.x = __fdividef(Fx1 - Fx2, gradient_delta.x);

                    t = point;
                    t.y += gradient_delta.y;
                    float Fy1 = interpolate(volume, buffer, t * voxel_size_inv);

                    t = point;
                    t.y -= gradient_delta.y;
                    float Fy2 = interpolate(volume, buffer, t * voxel_size_inv);

                    n.y = __fdividef(Fy1 - Fy2, gradient_delta.y);

                    t = point;
                    t.z += gradient_delta.z;
                    float Fz1 = interpolate(volume, buffer, t * voxel_size_inv);

                    t = point;
                    t.z -= gradient_delta.z;
                    float Fz2 = interpolate(volume, buffer, t * voxel_size_inv);

                    n.z = __fdividef(Fz1 - Fz2, gradient_delta.z);

                    n = normalized (aff.R * n);
                }

                output[idx] = make_float4(n.x, n.y, n.z, 0);
            }
        };

        __global__ void extract_normals_kernel (const ExtractNormals en, float4* output) { en(output); }
    }
}

size_t kfusion::device::extractCloud (const TsdfVolume& volume, const tsdf_buffer& buffer, const Aff3f& aff, PtrSz<Point> output)
{
    typedef FullScan6 FS;
    FS fs(volume);
    fs.aff = aff;

    dim3 block (FS::CTA_SIZE_X, FS::CTA_SIZE_Y);
    dim3 grid (divUp (volume.dims.x, block.x), divUp (volume.dims.y, block.y));

    extract_kernel<<<grid, block>>>(fs, buffer, output);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());

    int size;
    cudaSafeCall ( hipMemcpyFromSymbol(&size, HIP_SYMBOL(output_count), sizeof(size)) );
    return (size_t)size;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

size_t kfusion::device::extractSliceAsCloud (const TsdfVolume& volume, const kfusion::tsdf_buffer* buffer,
								const Vec3i minBounds, const Vec3i maxBounds, const Vec3i globalShift, const Aff3f& aff,
								PtrSz<Point> output)
{
	typedef FullScan6 FS;
    FS fs(volume);
    fs.aff = aff;

	dim3 block (FS::CTA_SIZE_X, FS::CTA_SIZE_Y);
    dim3 grid (divUp (volume.dims.x, block.x), divUp (volume.dims.y, block.y));

	//printf("buffer_origin: x: %d y: %d z: %d \n", buffer->origin_GRID.x, buffer->origin_GRID.y, buffer->origin_GRID.z);
	// Extraction call
	extractSliceKernel<<<grid, block>>>(fs, output, *buffer, minBounds, maxBounds, globalShift);

	cudaSafeCall ( hipGetLastError () );
	cudaSafeCall ( hipDeviceSynchronize () );

	int size;
	cudaSafeCall ( hipMemcpyFromSymbol(&size, HIP_SYMBOL(output_count), sizeof(size)) );
	return (size_t)size;
}

void kfusion::device::extractNormals (const TsdfVolume& volume, const tsdf_buffer& buffer, const PtrSz<Point>& points, const Aff3f& aff, const Mat3f& Rinv, float gradient_delta_factor, float4* output)
{
    ExtractNormals en(volume, buffer);
    en.points = points;
    en.gradient_delta = volume.voxel_size * gradient_delta_factor;
    en.aff = aff;
    en.Rinv = Rinv;

    dim3 block (256);
    dim3 grid (divUp ((int)points.size, block.x));

    extract_normals_kernel<<<grid, block>>>(en, output);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
kfusion::device::clearTSDFSlice (const TsdfVolume& volume, const kfusion::tsdf_buffer* buffer, const Vec3i offset)
{
	int newX = buffer->origin_GRID.x + offset.x;
	int newY = buffer->origin_GRID.y + offset.y;
	int3 minBounds, maxBounds;
	//X
	if(newX >= 0)
	{
		minBounds.x = buffer->origin_GRID.x;
		maxBounds.x = newX;
	}
	else
	{
		minBounds.x = newX + buffer->voxels_size.x;
		maxBounds.x = buffer->origin_GRID.x + buffer->voxels_size.x;
	}
	if(minBounds.x > maxBounds.x)
		std::swap(minBounds.x, maxBounds.x);
	//Y
	if(newY >= 0)
	{
		minBounds.y = buffer->origin_GRID.y;
		maxBounds.y = newY;
	}
	else
	{
		minBounds.y = newY + buffer->voxels_size.y;
		maxBounds.y = buffer->origin_GRID.y + buffer->voxels_size.y;
	}
	if(minBounds.y > maxBounds.y)
		std::swap(minBounds.y, maxBounds.y);
	//Z
	minBounds.z = buffer->origin_GRID.z;
	maxBounds.z = offset.z + 1;
	/*if(offset.x < 0)
	{
		minBounds.x += 1;
		maxBounds.x += 1;
	}
	if(offset.y < 0)
	{
		minBounds.y += 1;
		maxBounds.y += 1;
	}
	if(offset.z < 0)
	{
		minBounds.z += 1;
		maxBounds.z += 1;
	}*/


	//printf("clear minBounds: %d, %d, %d \n", minBounds.x, minBounds.y, minBounds.z);
	//printf("clear maxBounds: %d, %d, %d \n", maxBounds.x, maxBounds.y, maxBounds.z);
	// call kernel
	dim3 block (32, 16);
	dim3 grid (1, 1, 1);
	grid.x = divUp (buffer->voxels_size.x, block.x);
	grid.y = divUp (buffer->voxels_size.y, block.y);

	clearSliceKernel<<<grid, block>>>(volume, *buffer, minBounds, maxBounds);

	cudaSafeCall ( hipGetLastError () );
	cudaSafeCall (hipDeviceSynchronize ());
}
